#include "hip/hip_runtime.h"
#include "landauDeGennesLC.cuh"
#include "qTensorFunctions.h"
/*! \file landauDeGennesLC.cu */
/** @addtogroup forceKernels force Kernels
 * @{
 */

__device__ void gpu_get_six_neighbors(int3 &target, int &ixd, int &ixu,int &iyd, int &iyu,int &izd, int &izu,
                                      Index3D &latticeIndex, int3 &latticeSizes)
    {
    ixd = latticeIndex(wrap(target.x-1,latticeSizes.x),target.y,target.z);
    ixu = latticeIndex(wrap(target.x+1,latticeSizes.x),target.y,target.z);
    iyd = latticeIndex(target.x,wrap(target.y-1,latticeSizes.y),target.z);
    iyu = latticeIndex(target.x,wrap(target.y+1,latticeSizes.y),target.z);
    izd = latticeIndex(target.x,target.y,wrap(target.z-1,latticeSizes.z));
    izu = latticeIndex(target.x,target.y,wrap(target.z+1,latticeSizes.z));
    };

__device__ void gpu_phase_force(dVec &qCurrent, scalar &a, scalar &b, scalar &c, dVec &force)
    {
    force -= a*derivativeTrQ2(qCurrent);
    force -= b*derivativeTrQ3(qCurrent);
    force -= c*derivativeTrQ2Squared(qCurrent);
    //force += allPhaseComponentForces(qCurrent,a,b,c);
    }

__global__ void gpu_qTensor_computeBoundaryForcesGPU_kernel(dVec *d_force,
                                 dVec *d_spins,
                                 int *d_types,
                                 boundaryObject *d_bounds,
                                 Index3D latticeIndex,
                                 int N,
                                 bool zeroForce)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    int3 target = latticeIndex.inverseIndex(idx);
    int3 latticeSizes = latticeIndex.getSizes();
    dVec qCurrent, xDown, xUp, yDown,yUp,zDown,zUp;
    dVec force(0.0);
    dVec tempForce(0.0);
    if(d_types[idx] < 0) //compute only for sites adjacent to boundaries
        {
        qCurrent = d_spins[idx];
        //get neighbor indices and data
        int ixd, ixu,iyd,iyu,izd,izu;
        gpu_get_six_neighbors(target,ixd, ixu,iyd,iyu,izd,izu,latticeIndex,latticeSizes);

        if(d_types[ixd] > 0)
            {
            xDown = d_spins[ixd];
            computeBoundaryForce(qCurrent, xDown, d_bounds[d_types[ixd]-1],tempForce);
            force = force + tempForce;
            }
        if(d_types[ixu] > 0)
            {
            xUp = d_spins[ixu];
            computeBoundaryForce(qCurrent, xUp, d_bounds[d_types[ixu]-1],tempForce);
            force = force +tempForce;
            };
        if(d_types[iyd] > 0)
            {
            yDown = d_spins[iyd];
            computeBoundaryForce(qCurrent, yDown, d_bounds[d_types[iyd]-1],tempForce);
            force = force +tempForce;
            };
        if(d_types[iyu] > 0)
            {
            yUp = d_spins[iyu];
            computeBoundaryForce(qCurrent, yUp, d_bounds[d_types[iyu]-1],tempForce);
            force = force +tempForce;
            };
        if(d_types[izd] > 0)
            {
            zDown = d_spins[izd];
            computeBoundaryForce(qCurrent, zDown, d_bounds[d_types[izd]-1],tempForce);
            force = force +tempForce;
            };
        if(d_types[izu] > 0)
            {
            zUp = d_spins[izu];
            computeBoundaryForce(qCurrent, zUp, d_bounds[d_types[izu]-1],tempForce);
            force = force +tempForce;
            };
        };
    if(zeroForce)
        d_force[idx] = force;
    else
        d_force[idx] += force;
    }

__global__ void gpu_qTensor_firstDerivatives_kernel(cubicLatticeDerivativeVector *d_derivatives,
                                dVec *d_spins,
                                int *d_types,
                                Index3D latticeIndex,
                                int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    int3 target = latticeIndex.inverseIndex(idx);
    int3 latticeSizes = latticeIndex.getSizes();
    dVec qCurrent, xDown, xUp, yDown,yUp,zDown,zUp;
    cubicLatticeDerivativeVector zero(0.0);
    d_derivatives[idx] = zero;

    if(d_types[idx] <= 0) //no force on sites that are part of boundaries
        {
        //get neighbor indices and data
        int ixd, ixu,iyd,iyu,izd,izu;
        gpu_get_six_neighbors(target,ixd, ixu,iyd,iyu,izd,izu,latticeIndex,latticeSizes);
        xDown = d_spins[ixd];
        xUp = d_spins[ixu];
        yDown = d_spins[iyd];
        yUp = d_spins[iyu];
        zDown = d_spins[izd];
        zUp = d_spins[izu];
        if(d_types[idx] == 0) // bulk is easy
            {
            for (int qq = 0; qq < DIMENSION; ++qq)
                {
                d_derivatives[idx][qq] = 0.5*(xUp[qq]-xDown[qq]);
                };
            for (int qq = 0; qq < DIMENSION; ++qq)
                {
                d_derivatives[idx][DIMENSION+qq] = 0.5*(yUp[qq]-yDown[qq]);
                };
            for (int qq = 0; qq < DIMENSION; ++qq)
                {
                d_derivatives[idx][2*DIMENSION+qq] = 0.5*(zUp[qq]-zDown[qq]);
                };
            }
        else //near a boundary is less easy
            {
            if(d_types[ixd] <=0 &&d_types[ixu] <= 0) //x bulk
                {
                for (int qq = 0; qq < DIMENSION; ++qq)
                    {
                    d_derivatives[idx][qq] = 0.5*(xUp[qq]-xDown[qq]);
                    };
                }
            else if (d_types[ixu] > 0) //right is boundary
                {
                for (int qq = 0; qq < DIMENSION; ++qq)
                    {
                    d_derivatives[idx][qq] = (qCurrent[qq]-xDown[qq]);
                    };
                }
            else//left is boundary
                {
                for (int qq = 0; qq < DIMENSION; ++qq)
                    {
                    d_derivatives[idx][qq] = (xUp[qq]-qCurrent[qq]);
                    };
                };
            if(d_types[iyd] <=0 && d_types[iyu] <= 0) //y bulk
                {
                for (int qq = 0; qq < DIMENSION; ++qq)
                    {
                    d_derivatives[idx][DIMENSION+qq] = 0.5*(yUp[qq]-yDown[qq]);
                    };
                }
            else if (d_types[iyu] > 0) //up is boundary
                {
                for (int qq = 0; qq < DIMENSION; ++qq)
                    {
                    d_derivatives[idx][DIMENSION+qq] = (qCurrent[qq]-yDown[qq]);
                    };
                }
            else//down is boundary
                {
                for (int qq = 0; qq < DIMENSION; ++qq)
                    {
                    d_derivatives[idx][DIMENSION+qq] = (yUp[qq]-qCurrent[qq]);
                    };
                };
            if(d_types[izd] <=0 && d_types[izu] <= 0) //z bulk
                {
                for (int qq = 0; qq < DIMENSION; ++qq)
                    {
                    d_derivatives[idx][2*DIMENSION+qq] = 0.5*(zUp[qq]-zDown[qq]);
                    };
                }
            else if (d_types[izu] > 0) //up is boundary
                {
                for (int qq = 0; qq < DIMENSION; ++qq)
                    {
                    d_derivatives[idx][2*DIMENSION+qq] = (qCurrent[qq]-zDown[qq]);
                    };
                }
            else//down is boundary
                {
                for (int qq = 0; qq < DIMENSION; ++qq)
                    {
                    d_derivatives[idx][2*DIMENSION+qq] = (zUp[qq]-qCurrent[qq]);
                    };
                };
            };
        };
    }

__global__ void gpu_qTensor_oneConstantForce_kernel(dVec *d_force,
                                dVec *d_spins,
                                int *d_types,
                                int *d_latticeNeighbors,
                                Index3D latticeIndex,
                                Index2D neighborIndex,
                                scalar a,scalar b,scalar c,scalar L1,
                                int N,
                                bool zeroForce)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    int3 target = latticeIndex.inverseIndex(idx);
    int3 latticeSizes = latticeIndex.getSizes();
    dVec qCurrent, xDown, xUp, yDown,yUp,zDown,zUp;
    dVec force(0.0);

    if(d_types[idx] <= 0) //no force on sites that are part of boundaries
        {
        //phase part is simple
        qCurrent = d_spins[idx];
        gpu_phase_force(qCurrent, a, b, c, force);

        //get neighbor indices and data
        int ixd, ixu,iyd,iyu,izd,izu;
        ixd =d_latticeNeighbors[neighborIndex(0,idx)];
        ixu =d_latticeNeighbors[neighborIndex(1,idx)];
        iyd =d_latticeNeighbors[neighborIndex(2,idx)];
        iyu =d_latticeNeighbors[neighborIndex(3,idx)];
        izd =d_latticeNeighbors[neighborIndex(4,idx)];
        izu =d_latticeNeighbors[neighborIndex(5,idx)];

        xDown = d_spins[ixd]; xUp = d_spins[ixu];
        yDown = d_spins[iyd]; yUp = d_spins[iyu];
        zDown = d_spins[izd]; zUp = d_spins[izu];
        dVec spatialTerm(0.0);
        if(d_types[idx] == 0) // bulk is easy
            {
            spatialTerm = L1*(6.0*qCurrent-xDown-xUp-yDown-yUp-zDown-zUp);
            scalar AxxAyy = spatialTerm[0]+spatialTerm[3];
            spatialTerm[0] += AxxAyy;
            spatialTerm[1] *= 2.0;
            spatialTerm[2] *= 2.0;
            spatialTerm[3] += AxxAyy;
            spatialTerm[4] *= 2.0;
            }
        else //near a boundary is less easy... ternary operators are slightly better than many ifs (particularly if boundaries are typically jagged)
            {
            if(d_types[ixd]>0)//xDown is a boundary
                {
                spatialTerm -= (xUp-qCurrent);
                }
            if(d_types[ixu] >0)//xUp is a boundary
                {
                spatialTerm -= (xDown-qCurrent);//negative derivative and negative nu_x cancel
                }
            if(d_types[iyd]>1)//ydown
                {
                spatialTerm -= (yUp-qCurrent);
                }
            if(d_types[iyu] >0)
                {
                spatialTerm -= (yDown-qCurrent);//negative derivative and negative nu_y cancel
                }
            if(d_types[izd] >0)//zDown is boundary
                {
                spatialTerm -= (zUp-qCurrent);
                }
            if(d_types[izu] >0)
                {
                spatialTerm -= (zDown-qCurrent);//negative derivative and negative nu_z cancel
                }
            scalar crossTerm = spatialTerm[0]+spatialTerm[3];
            spatialTerm[0] += crossTerm;
            spatialTerm[1] *= 2.0;
            spatialTerm[2] *= 2.0;
            spatialTerm[3] += crossTerm;
            spatialTerm[4] *= 2.0;
            };
        force -= L1*spatialTerm;
        };
    if(zeroForce)
        d_force[idx] = force;
    else
        d_force[idx] += force;
    }

__global__ void gpu_qTensor_twoConstantForce_kernel(dVec *d_force,
                                dVec *d_spins,
                                int *d_types,
                                cubicLatticeDerivativeVector *d_derivatives,
                                Index3D latticeIndex,
                                scalar a,scalar b,scalar c,scalar L1,scalar L2, scalar q0,
                                int N,
                                bool zeroForce)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    int3 target = latticeIndex.inverseIndex(idx);
    int3 latticeSizes = latticeIndex.getSizes();
    dVec qCurrent, xDown, xUp, yDown,yUp,zDown,zUp;
    dVec force(0.0);

    if(d_types[idx] <= 0) //no force on sites that are part of boundaries
        {
        //phase part is simple
        qCurrent = d_spins[idx];
        gpu_phase_force(qCurrent, a, b, c, force);

        //get neighbor indices and data
        int ixd, ixu,iyd,iyu,izd,izu;
        gpu_get_six_neighbors(target,ixd, ixu,iyd,iyu,izd,izu,latticeIndex,latticeSizes);
        xDown = d_spins[ixd]; xUp = d_spins[ixu];
        yDown = d_spins[iyd]; yUp = d_spins[iyu];
        zDown = d_spins[izd]; zUp = d_spins[izu];
        cubicLatticeDerivativeVector qCurrentDerivative = d_derivatives[idx];
        cubicLatticeDerivativeVector xDownDerivative = d_derivatives[ixd];
        cubicLatticeDerivativeVector xUpDerivative = d_derivatives[ixu];
        cubicLatticeDerivativeVector yDownDerivative = d_derivatives[iyd];
        cubicLatticeDerivativeVector yUpDerivative = d_derivatives[iyu];
        cubicLatticeDerivativeVector zDownDerivative = d_derivatives[izd];
        cubicLatticeDerivativeVector zUpDerivative = d_derivatives[izu];

        dVec xMinusTerm(0.0);
        dVec xPlusTerm(0.0);
        dVec yMinusTerm(0.0);
        dVec yPlusTerm(0.0);
        dVec zMinusTerm(0.0);
        dVec zPlusTerm(0.0);
        if(d_types[ixd] <= 0) //xMinus
            {
            xMinusTerm[0]=-(L1*(2*qCurrent[0] + 2*qCurrent[3] + 324*(q0*q0)*(2*qCurrent[0] + qCurrent[3]) + qCurrentDerivative[12] - 2*xDown[0] - 2*xDown[3] - 18*q0*(qCurrentDerivative[11] - 2*(qCurrentDerivative[7] + xDown[4])) + xDownDerivative[12]))/2. - (L2*(2*qCurrent[0] + qCurrentDerivative[6] + qCurrentDerivative[12] - 2*xDown[0] + xDownDerivative[6] + xDownDerivative[12]))/2.;

            xMinusTerm[1]=(L1*(qCurrentDerivative[5] - 2*(qCurrent[1] + 324*(q0*q0)*qCurrent[1] - xDown[1] + 9*q0*(qCurrentDerivative[9] + qCurrentDerivative[10] - qCurrentDerivative[13] + 2*xDown[2])) + xDownDerivative[5]))/2. - (L2*(2*qCurrent[1] + qCurrentDerivative[8] + qCurrentDerivative[14] - 2*xDown[1] + xDownDerivative[8] + xDownDerivative[14]))/2.;

            xMinusTerm[2]=-(L1*(1296*(q0*q0)*qCurrent[2] - 36*q0*(2*qCurrentDerivative[5] + qCurrentDerivative[8] + qCurrentDerivative[14] + 2*xDown[1]) - 2*(-2*qCurrent[2] + qCurrentDerivative[10] + 2*xDown[2] + xDownDerivative[10])))/4. + (L2*(-2*qCurrent[2] - qCurrentDerivative[9] + qCurrentDerivative[10] + qCurrentDerivative[13] + 2*xDown[2] - xDownDerivative[9] + xDownDerivative[10] + xDownDerivative[13]))/2.;

            xMinusTerm[3]=-(L1*(2*qCurrent[0] + 324*(q0*q0)*qCurrent[0] + 4*qCurrent[3] + 648*(q0*q0)*qCurrent[3] - qCurrentDerivative[6] + 18*q0*qCurrentDerivative[7] + 18*q0*qCurrentDerivative[11] + qCurrentDerivative[12] - 2*xDown[0] - 4*xDown[3] + 72*q0*xDown[4] - xDownDerivative[6] + xDownDerivative[12]))/2.;

            xMinusTerm[4]=(L1*(-4*qCurrent[4] - 648*(q0*q0)*qCurrent[4] + qCurrentDerivative[7] + qCurrentDerivative[11] + 18*q0*(qCurrentDerivative[6] - qCurrentDerivative[12] + 2*xDown[0] + 4*xDown[3]) + 4*xDown[4] + xDownDerivative[7] + xDownDerivative[11]))/2.;
            }
        if(d_types[ixu] <= 0) //xPlus
            {
            xPlusTerm[0]=(L2*(-2*qCurrent[0] + qCurrentDerivative[6] + qCurrentDerivative[12] + 2*xUp[0] + xUpDerivative[6] + xUpDerivative[12]))/2. - (L1*(648*(q0*q0)*(2*qCurrent[0] + qCurrent[3]) - 36*q0*(-2*qCurrentDerivative[7] + qCurrentDerivative[11] + 2*xUp[4]) - 2*(-2*qCurrent[0] - 2*qCurrent[3] + qCurrentDerivative[12] + 2*xUp[0] + 2*xUp[3] + xUpDerivative[12])))/4.;

            xPlusTerm[1]=-(L1*(qCurrentDerivative[5] + 2*(qCurrent[1] + 324*(q0*q0)*qCurrent[1] - xUp[1] + 9*q0*(qCurrentDerivative[9] + qCurrentDerivative[10] - qCurrentDerivative[13] - 2*xUp[2])) + xUpDerivative[5]))/2. + (L2*(-2*qCurrent[1] + qCurrentDerivative[8] + qCurrentDerivative[14] + 2*xUp[1] + xUpDerivative[8] + xUpDerivative[14]))/2.;

            xPlusTerm[2]=-(L1*(2*qCurrent[2] + 648*(q0*q0)*qCurrent[2] + qCurrentDerivative[10] - 18*q0*(2*qCurrentDerivative[5] + qCurrentDerivative[8] + qCurrentDerivative[14] - 2*xUp[1]) - 2*xUp[2] + xUpDerivative[10]))/2. - (L2*(2*qCurrent[2] - qCurrentDerivative[9] + qCurrentDerivative[10] + qCurrentDerivative[13] - 2*xUp[2] - xUpDerivative[9] + xUpDerivative[10] + xUpDerivative[13]))/2.;

            xPlusTerm[3]=-(L1*(2*qCurrent[0] + 324*(q0*q0)*qCurrent[0] + 4*qCurrent[3] + 648*(q0*q0)*qCurrent[3] + qCurrentDerivative[6] + 18*q0*qCurrentDerivative[7] + 18*q0*qCurrentDerivative[11] - qCurrentDerivative[12] - 2*xUp[0] - 4*xUp[3] - 72*q0*xUp[4] + xUpDerivative[6] - xUpDerivative[12]))/2.;

            xPlusTerm[4]=-(L1*(4*qCurrent[4] + 648*(q0*q0)*qCurrent[4] + qCurrentDerivative[7] + qCurrentDerivative[11] - 18*q0*(qCurrentDerivative[6] - qCurrentDerivative[12] - 2*xUp[0] - 4*xUp[3]) - 4*xUp[4] + xUpDerivative[7] + xUpDerivative[11]))/2.;
            }

        if(d_types[iyd] <= 0) //yMinus
            {
            yMinusTerm[0]=-(L1*(4*qCurrent[0] + 648*(q0*q0)*qCurrent[0] + 2*qCurrent[3] + 324*(q0*q0)*qCurrent[3] - qCurrentDerivative[1] - 18*q0*qCurrentDerivative[4] - 18*q0*qCurrentDerivative[11] + qCurrentDerivative[14] - 4*yDown[0] - 72*q0*yDown[2] - 2*yDown[3] - yDownDerivative[1] + yDownDerivative[14]))/2.;

            yMinusTerm[1]=-(L1*(1296*(q0*q0)*qCurrent[1] + 36*q0*(-qCurrentDerivative[2] + qCurrentDerivative[10] - qCurrentDerivative[13] - 2*yDown[4]) - 2*(-2*qCurrent[1] + qCurrentDerivative[3] + 2*yDown[1] + yDownDerivative[3])))/4. - (L2*(2*qCurrent[1] + qCurrentDerivative[0] + qCurrentDerivative[12] - 2*yDown[1] + yDownDerivative[0] + yDownDerivative[12]))/2.;

            yMinusTerm[2]=(L1*(-4*qCurrent[2] - 648*(q0*q0)*qCurrent[2] + qCurrentDerivative[4] + qCurrentDerivative[11] + 4*yDown[2] - 18*q0*(qCurrentDerivative[1] - qCurrentDerivative[14] + 4*yDown[0] + 2*yDown[3]) + yDownDerivative[4] + yDownDerivative[11]))/2.;

            yMinusTerm[3]=-(L1*(2*qCurrent[0] + 2*qCurrent[3] + 324*(q0*q0)*(qCurrent[0] + 2*qCurrent[3]) + qCurrentDerivative[14] - 2*yDown[0] + 18*q0*(qCurrentDerivative[11] - 2*(qCurrentDerivative[4] + yDown[2])) - 2*yDown[3] + yDownDerivative[14]))/2. - (L2*(2*qCurrent[3] + qCurrentDerivative[1] + qCurrentDerivative[14] - 2*yDown[3] + yDownDerivative[1] + yDownDerivative[14]));

            yMinusTerm[4]=(L2*(-2*qCurrent[4] - qCurrentDerivative[2] + qCurrentDerivative[10] + qCurrentDerivative[13] + 2*yDown[4] - yDownDerivative[2] + yDownDerivative[10] + yDownDerivative[13]))/2. - (L1*(1296*(q0*q0)*qCurrent[4] + 36*q0*(qCurrentDerivative[0] + qCurrentDerivative[12] + 2*(qCurrentDerivative[3] + yDown[1])) - 2*(-2*qCurrent[4] + qCurrentDerivative[13] + 2*yDown[4] + yDownDerivative[13])))/4.;
            }

        if(d_types[iyu] <= 0) //yPlus
            {
            yPlusTerm[0]=-(L1*(4*qCurrent[0] + 648*(q0*q0)*qCurrent[0] + 2*qCurrent[3] + 324*(q0*q0)*qCurrent[3] + qCurrentDerivative[1] - 18*q0*qCurrentDerivative[4] - 18*q0*qCurrentDerivative[11] - qCurrentDerivative[14] - 4*yUp[0] + 72*q0*yUp[2] - 2*yUp[3] + yUpDerivative[1] - yUpDerivative[14]))/2.;

            yPlusTerm[1]=-(L1*(2*qCurrent[1] + 648*(q0*q0)*qCurrent[1] + qCurrentDerivative[3] - 2*yUp[1] + 18*q0*(-qCurrentDerivative[2] + qCurrentDerivative[10] - qCurrentDerivative[13] + 2*yUp[4]) + yUpDerivative[3]))/2. + (L2*(-2*qCurrent[1] + qCurrentDerivative[0] + qCurrentDerivative[12] + 2*yUp[1] + yUpDerivative[0] + yUpDerivative[12]))/2.;

            yPlusTerm[2]=-(L1*(4*qCurrent[2] + 648*(q0*q0)*qCurrent[2] + qCurrentDerivative[4] + qCurrentDerivative[11] - 4*yUp[2] + 18*q0*(qCurrentDerivative[1] - qCurrentDerivative[14] - 4*yUp[0] - 2*yUp[3]) + yUpDerivative[4] + yUpDerivative[11]))/2.;

            yPlusTerm[3]=(L2*(-2*qCurrent[3] + qCurrentDerivative[1] + qCurrentDerivative[14] + 2*yUp[3] + yUpDerivative[1] + yUpDerivative[14]))/2. - (L1*(648*(q0*q0)*(qCurrent[0] + 2*qCurrent[3]) + 36*q0*(-2*qCurrentDerivative[4] + qCurrentDerivative[11] + 2*yUp[2]) - 2*(-2*qCurrent[0] - 2*qCurrent[3] + qCurrentDerivative[14] + 2*yUp[0] + 2*yUp[3] + yUpDerivative[14])))/4.;

            yPlusTerm[4]=-(L1*(2*qCurrent[4] + 648*(q0*q0)*qCurrent[4] + qCurrentDerivative[13] + 18*q0*(qCurrentDerivative[0] + 2*qCurrentDerivative[3] + qCurrentDerivative[12] - 2*yUp[1]) - 2*yUp[4] + yUpDerivative[13]))/2. - (L2*(2*qCurrent[4] - qCurrentDerivative[2] + qCurrentDerivative[10] + qCurrentDerivative[13] - 2*yUp[4] - yUpDerivative[2] + yUpDerivative[10] + yUpDerivative[13]))/2.;
            }

        if(d_types[izd] <= 0) //zMinus
            {
            zMinusTerm[0]=(L1*(qCurrentDerivative[2] - 2*(qCurrent[0] + 162*(q0*q0)*(2*qCurrent[0] + qCurrent[3]) - zDown[0] + 9*q0*(-qCurrentDerivative[4] + 2*qCurrentDerivative[7] + 2*zDown[1])) + zDownDerivative[2]))/2. + (L2*(-2*qCurrent[0] - 2*qCurrent[3] + qCurrentDerivative[2] + qCurrentDerivative[9] + 2*zDown[0] + 2*zDown[3] + zDownDerivative[2] + zDownDerivative[9]))/2.;

            zMinusTerm[1]=-(L1*(1296*(q0*q0)*qCurrent[1] - 36*q0*(qCurrentDerivative[2] - qCurrentDerivative[9] + 2*zDown[0] - 2*zDown[3]) - 2*(-4*qCurrent[1] + qCurrentDerivative[4] + qCurrentDerivative[7] + 4*zDown[1] + zDownDerivative[4] + zDownDerivative[7])))/4.;

            zMinusTerm[2]=-(L1*(2*qCurrent[2] + 648*(q0*q0)*qCurrent[2] + qCurrentDerivative[0] + 18*q0*qCurrentDerivative[1] + qCurrentDerivative[3] - 36*q0*qCurrentDerivative[5] - 18*q0*qCurrentDerivative[8] - 2*zDown[2] + 36*q0*zDown[4] + zDownDerivative[0] + zDownDerivative[3]))/2. - (L2*(2*qCurrent[2] + qCurrentDerivative[0] + qCurrentDerivative[6] - 2*zDown[2] + zDownDerivative[0] + zDownDerivative[6]))/2.;

            zMinusTerm[3]=(L2*(-2*qCurrent[0] - 2*qCurrent[3] + qCurrentDerivative[2] + qCurrentDerivative[9] + 2*zDown[0] + 2*zDown[3] + zDownDerivative[2] + zDownDerivative[9]))/2. - (L1*(648*(q0*q0)*(qCurrent[0] + 2*qCurrent[3]) + 36*q0*(qCurrentDerivative[7] - 2*(qCurrentDerivative[4] + zDown[1])) - 2*(-2*qCurrent[3] + qCurrentDerivative[9] + 2*zDown[3] + zDownDerivative[9])))/4.;

            zMinusTerm[4]=-(L2*(2*qCurrent[4] + qCurrentDerivative[1] + qCurrentDerivative[8] - 2*zDown[4] + zDownDerivative[1] + zDownDerivative[8]))/2. - (L1*(2*qCurrent[4] + 648*(q0*q0)*qCurrent[4] + qCurrentDerivative[5] + qCurrentDerivative[8] + 18*q0*(qCurrentDerivative[0] + 2*qCurrentDerivative[3] - qCurrentDerivative[6] - 2*zDown[2]) - 2*zDown[4] + zDownDerivative[5] + zDownDerivative[8]))/2.;
            }

        if(d_types[izu] <= 0) //zPlus
            {
            zPlusTerm[0]=-(L1*(qCurrentDerivative[2] + 2*(qCurrent[0] + 324*(q0*q0)*qCurrent[0] + 162*(q0*q0)*qCurrent[3] - 9*q0*qCurrentDerivative[4] + 18*q0*qCurrentDerivative[7] - zUp[0] - 18*q0*zUp[1]) + zUpDerivative[2]))/2. - (L2*(2*qCurrent[0] + 2*qCurrent[3] + qCurrentDerivative[2] + qCurrentDerivative[9] - 2*zUp[0] - 2*zUp[3] + zUpDerivative[2] + zUpDerivative[9]))/2.;

            zPlusTerm[1]=-(L1*(4*qCurrent[1] + 648*(q0*q0)*qCurrent[1] + qCurrentDerivative[4] + qCurrentDerivative[7] - 4*zUp[1] - 18*q0*(qCurrentDerivative[2] - qCurrentDerivative[9] - 2*zUp[0] + 2*zUp[3]) + zUpDerivative[4] + zUpDerivative[7]))/2.;

            zPlusTerm[2]=(L1*(-2*qCurrent[2] - 648*(q0*q0)*qCurrent[2] + qCurrentDerivative[0] - 18*q0*qCurrentDerivative[1] + qCurrentDerivative[3] + 36*q0*qCurrentDerivative[5] + 18*q0*qCurrentDerivative[8] + 2*zUp[2] + 36*q0*zUp[4] + zUpDerivative[0] + zUpDerivative[3]))/2. + (L2*(-2*qCurrent[2] + qCurrentDerivative[0] + qCurrentDerivative[6] + 2*zUp[2] + zUpDerivative[0] + zUpDerivative[6]))/2.;

            zPlusTerm[3]=-(L1*(2*qCurrent[3] + 324*(q0*q0)*(qCurrent[0] + 2*qCurrent[3]) + qCurrentDerivative[9] + 18*q0*(-2*qCurrentDerivative[4] + qCurrentDerivative[7] + 2*zUp[1]) - 2*zUp[3] + zUpDerivative[9]))/2. - (L2*(2*qCurrent[0] + 2*qCurrent[3] + qCurrentDerivative[2] + qCurrentDerivative[9] - 2*zUp[0] - 2*zUp[3] + zUpDerivative[2] + zUpDerivative[9]))/2.;

            zPlusTerm[4]=(L2*(-2*qCurrent[4] + qCurrentDerivative[1] + qCurrentDerivative[8] + 2*zUp[4] + zUpDerivative[1] + zUpDerivative[8]))/2. - (L1*(1296*(q0*q0)*qCurrent[4] + 36*q0*(qCurrentDerivative[0] - qCurrentDerivative[6] + 2*(qCurrentDerivative[3] + zUp[2])) - 2*(-2*qCurrent[4] + qCurrentDerivative[5] + qCurrentDerivative[8] + 2*zUp[4] + zUpDerivative[5] + zUpDerivative[8])))/4.;
            }

        force += xMinusTerm+xPlusTerm+yMinusTerm+yPlusTerm+zMinusTerm+zPlusTerm;

        };
    if(zeroForce)
        d_force[idx] = force;
    else
        d_force[idx] += force;
    }

__global__ void gpu_qTensor_threeConstantForce_kernel(dVec *d_force,
                                dVec *d_spins,
                                int *d_types,
                                cubicLatticeDerivativeVector *d_derivatives,
                                Index3D latticeIndex,
                                scalar a,scalar b,scalar c,scalar L1,scalar L2, scalar L3,
                                int N,
                                bool zeroForce)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    int3 target = latticeIndex.inverseIndex(idx);
    int3 latticeSizes = latticeIndex.getSizes();
    dVec qCurrent, xDown, xUp, yDown,yUp,zDown,zUp;
    dVec force(0.0);

    if(d_types[idx] <= 0) //no force on sites that are part of boundaries
        {
        //phase part is simple
        qCurrent = d_spins[idx];
        gpu_phase_force(qCurrent, a, b, c, force);

        //get neighbor indices and data
        int ixd, ixu,iyd,iyu,izd,izu;
        gpu_get_six_neighbors(target,ixd, ixu,iyd,iyu,izd,izu,latticeIndex,latticeSizes);
        xDown = d_spins[ixd]; xUp = d_spins[ixu];
        yDown = d_spins[iyd]; yUp = d_spins[iyu];
        zDown = d_spins[izd]; zUp = d_spins[izu];
        cubicLatticeDerivativeVector qCurrentDerivative = d_derivatives[idx];
        cubicLatticeDerivativeVector xDownDerivative = d_derivatives[ixd];
        cubicLatticeDerivativeVector xUpDerivative = d_derivatives[ixu];
        cubicLatticeDerivativeVector yDownDerivative = d_derivatives[iyd];
        cubicLatticeDerivativeVector yUpDerivative = d_derivatives[iyu];
        cubicLatticeDerivativeVector zDownDerivative = d_derivatives[izd];
        cubicLatticeDerivativeVector zUpDerivative = d_derivatives[izu];

        dVec xMinusTerm(0.0);
        dVec xPlusTerm(0.0);
        dVec yMinusTerm(0.0);
        dVec yPlusTerm(0.0);
        dVec zMinusTerm(0.0);
        dVec zPlusTerm(0.0);
        if(d_types[ixd] <= 0) //xMinus
            {
            xMinusTerm[0]=-(L1*(8*qCurrent[0] + 4*qCurrent[3] - 8*xDown[0] - 4*xDown[3]))/4. - (L2*(2*qCurrent[0] + qCurrentDerivative[6] + qCurrentDerivative[12] - 2*xDown[0] + xDownDerivative[6] + xDownDerivative[12]))/2. - (L3*(3*(qCurrent[0]*qCurrent[0]) + qCurrent[1]*qCurrent[1] + qCurrent[2]*qCurrent[2] + qCurrent[3]*qCurrent[3] + qCurrent[4]*qCurrent[4] - qCurrentDerivative[10]*qCurrentDerivative[10] - qCurrentDerivative[11]*qCurrentDerivative[11] - qCurrentDerivative[12]*qCurrentDerivative[12] - qCurrentDerivative[13]*qCurrentDerivative[13] - qCurrentDerivative[14]*qCurrentDerivative[14] - xDown[0]*xDown[0] + xDown[1]*xDown[1] + xDown[2]*xDown[2] + xDown[3]*xDown[3] + xDown[4]*xDown[4] + 2*qCurrent[0]*qCurrent[3] + 2*qCurrent[1]*qCurrentDerivative[5] + qCurrent[1]*qCurrentDerivative[8] + qCurrent[2]*qCurrentDerivative[13] - qCurrentDerivative[10]*(-2*qCurrent[2] + qCurrentDerivative[13]) - 2*qCurrent[0]*xDown[0] - 2*qCurrent[1]*xDown[1] - 2*qCurrent[2]*xDown[2] - 2*qCurrent[0]*xDown[3] - 2*qCurrent[3]*xDown[3] - 2*qCurrent[4]*xDown[4] + 2*xDown[1]*xDownDerivative[5] + xDown[1]*xDownDerivative[8] + 2*xDown[2]*xDownDerivative[10] + xDown[2]*xDownDerivative[13]))/2.;

            xMinusTerm[1]=-2*L1*(qCurrent[1] - xDown[1]) - (L3*(qCurrentDerivative[8]*(qCurrent[0] + 2*qCurrent[3] - xDown[0] - 2*xDown[3]) + qCurrentDerivative[5]*(2*qCurrent[0] + qCurrent[3] - 2*xDown[0] - xDown[3]) + 2*(qCurrent[0]*qCurrent[1] + qCurrent[2]*qCurrentDerivative[7] + qCurrent[4]*qCurrentDerivative[9] + qCurrent[2]*qCurrentDerivative[11] + qCurrent[1]*xDown[0] - qCurrent[0]*xDown[1] - xDown[0]*xDown[1] - qCurrentDerivative[6]*(-2*qCurrent[1] + xDown[1]) - qCurrentDerivative[7]*xDown[2] - qCurrentDerivative[9]*xDown[4] + xDown[1]*xDownDerivative[6] + xDown[2]*xDownDerivative[11])))/2. - (L2*(2*qCurrent[1] + qCurrentDerivative[8] + qCurrentDerivative[14] - 2*xDown[1] + xDownDerivative[8] + xDownDerivative[14]))/2.;

            xMinusTerm[2]=-2*L1*(qCurrent[2] - xDown[2]) - (L3*(qCurrentDerivative[13]*(qCurrent[0] + 2*qCurrent[3] - xDown[0] - 2*xDown[3]) + qCurrentDerivative[10]*(2*qCurrent[0] + qCurrent[3] - 2*xDown[0] - xDown[3]) + 2*(qCurrent[0]*qCurrent[2] + qCurrent[1]*qCurrentDerivative[7] + 2*qCurrent[2]*qCurrentDerivative[12] + qCurrent[4]*qCurrentDerivative[14] + qCurrent[2]*xDown[0] + qCurrentDerivative[11]*(qCurrent[1] - xDown[1]) - qCurrent[0]*xDown[2] - qCurrentDerivative[12]*xDown[2] - xDown[0]*xDown[2] - qCurrentDerivative[14]*xDown[4] + xDown[1]*xDownDerivative[7] + xDown[2]*xDownDerivative[12])))/2. + (L2*(-2*qCurrent[2] - qCurrentDerivative[9] + qCurrentDerivative[10] + qCurrentDerivative[13] + 2*xDown[2] - xDownDerivative[9] + xDownDerivative[10] + xDownDerivative[13]))/2.;

            xMinusTerm[3]=-(L1*(4*qCurrent[0] + 8*qCurrent[3] - 4*xDown[0] - 8*xDown[3]))/4. - (L3*(qCurrent[0]*qCurrent[0] + qCurrentDerivative[5]*qCurrentDerivative[5] + qCurrentDerivative[6]*qCurrentDerivative[6] + qCurrentDerivative[7]*qCurrentDerivative[7] + qCurrentDerivative[8]*qCurrentDerivative[8] + qCurrentDerivative[9]*qCurrentDerivative[9] - qCurrentDerivative[10]*qCurrentDerivative[10] - qCurrentDerivative[11]*qCurrentDerivative[11] - qCurrentDerivative[12]*qCurrentDerivative[12] - qCurrentDerivative[13]*qCurrentDerivative[13] - qCurrentDerivative[14]*qCurrentDerivative[14] - xDown[0]*xDown[0] + 2*qCurrent[0]*qCurrent[3] + 2*qCurrent[1]*qCurrentDerivative[8] + qCurrentDerivative[5]*(qCurrent[1] + qCurrentDerivative[8]) + qCurrentDerivative[10]*(qCurrent[2] - qCurrentDerivative[13]) + 2*qCurrent[2]*qCurrentDerivative[13] + 2*qCurrent[3]*xDown[0] - 2*qCurrent[0]*xDown[3] - 2*xDown[0]*xDown[3] + xDown[1]*xDownDerivative[5] + 2*xDown[1]*xDownDerivative[8] + xDown[2]*xDownDerivative[10] + 2*xDown[2]*xDownDerivative[13]))/2.;

            xMinusTerm[4]=-2*L1*(qCurrent[4] - xDown[4]) - (L3*(2*qCurrent[1]*qCurrentDerivative[9] + 2*qCurrentDerivative[5]*qCurrentDerivative[10] + qCurrentDerivative[8]*qCurrentDerivative[10] + 2*qCurrentDerivative[6]*qCurrentDerivative[11] + 2*qCurrentDerivative[7]*qCurrentDerivative[12] + qCurrentDerivative[5]*qCurrentDerivative[13] + 2*qCurrentDerivative[8]*qCurrentDerivative[13] + 2*qCurrent[2]*qCurrentDerivative[14] + 2*qCurrentDerivative[9]*qCurrentDerivative[14] + 2*qCurrent[0]*(qCurrent[4] - xDown[4]) + 2*xDown[0]*(qCurrent[4] - xDown[4]) + 2*xDown[1]*xDownDerivative[9] + 2*xDown[2]*xDownDerivative[14]))/2.;
            }
        if(d_types[ixu] <= 0) //xPlus
            {
            xPlusTerm[0]=-(L1*(2*qCurrent[0] + qCurrent[3] - 2*xUp[0] - xUp[3])) + (L2*(-2*qCurrent[0] + qCurrentDerivative[6] + qCurrentDerivative[12] + 2*xUp[0] + xUpDerivative[6] + xUpDerivative[12]))/2. + (L3*(-3*(qCurrent[0]*qCurrent[0]) - qCurrent[1]*qCurrent[1] - qCurrent[2]*qCurrent[2] - qCurrent[3]*qCurrent[3] - qCurrent[4]*qCurrent[4] + qCurrentDerivative[10]*qCurrentDerivative[10] + qCurrentDerivative[11]*qCurrentDerivative[11] + qCurrentDerivative[12]*qCurrentDerivative[12] + qCurrentDerivative[13]*qCurrentDerivative[13] + qCurrentDerivative[14]*qCurrentDerivative[14] + xUp[0]*xUp[0] - xUp[1]*xUp[1] - xUp[2]*xUp[2] - xUp[3]*xUp[3] - xUp[4]*xUp[4] - 2*qCurrent[0]*qCurrent[3] + 2*qCurrent[1]*qCurrentDerivative[5] + qCurrent[1]*qCurrentDerivative[8] + qCurrent[2]*qCurrentDerivative[13] + qCurrentDerivative[10]*(2*qCurrent[2] + qCurrentDerivative[13]) + 2*qCurrent[0]*xUp[0] + 2*qCurrent[1]*xUp[1] + 2*qCurrent[2]*xUp[2] + 2*qCurrent[0]*xUp[3] + 2*qCurrent[3]*xUp[3] + 2*qCurrent[4]*xUp[4] + 2*xUp[1]*xUpDerivative[5] + xUp[1]*xUpDerivative[8] + 2*xUp[2]*xUpDerivative[10] + xUp[2]*xUpDerivative[13]))/2.;

            xPlusTerm[1]=-2*L1*(qCurrent[1] - xUp[1]) - (L3*(qCurrentDerivative[5]*(-2*qCurrent[0] - qCurrent[3] + 2*xUp[0] + xUp[3]) + qCurrentDerivative[8]*(-qCurrent[0] - 2*qCurrent[3] + xUp[0] + 2*xUp[3]) - 2*(qCurrent[2]*qCurrentDerivative[7] + qCurrent[4]*qCurrentDerivative[9] + qCurrent[2]*qCurrentDerivative[11] - qCurrent[1]*xUp[0] + qCurrentDerivative[6]*(2*qCurrent[1] - xUp[1]) + xUp[0]*xUp[1] + qCurrent[0]*(-qCurrent[1] + xUp[1]) - qCurrentDerivative[7]*xUp[2] - qCurrentDerivative[9]*xUp[4] + xUp[1]*xUpDerivative[6] + xUp[2]*xUpDerivative[11])))/2. + (L2*(-2*qCurrent[1] + qCurrentDerivative[8] + qCurrentDerivative[14] + 2*xUp[1] + xUpDerivative[8] + xUpDerivative[14]))/2.;

            xPlusTerm[2]=-2*L1*(qCurrent[2] - xUp[2]) - (L3*(qCurrentDerivative[10]*(-2*qCurrent[0] - qCurrent[3] + 2*xUp[0] + xUp[3]) + qCurrentDerivative[13]*(-qCurrent[0] - 2*qCurrent[3] + xUp[0] + 2*xUp[3]) - 2*(-(qCurrent[0]*qCurrent[2]) + qCurrent[1]*qCurrentDerivative[7] + 2*qCurrent[2]*qCurrentDerivative[12] + qCurrent[4]*qCurrentDerivative[14] - qCurrent[2]*xUp[0] + qCurrentDerivative[11]*(qCurrent[1] - xUp[1]) + qCurrent[0]*xUp[2] - qCurrentDerivative[12]*xUp[2] + xUp[0]*xUp[2] - qCurrentDerivative[14]*xUp[4] + xUp[1]*xUpDerivative[7] + xUp[2]*xUpDerivative[12])))/2. - (L2*(2*qCurrent[2] - qCurrentDerivative[9] + qCurrentDerivative[10] + qCurrentDerivative[13] - 2*xUp[2] - xUpDerivative[9] + xUpDerivative[10] + xUpDerivative[13]))/2.;

            xPlusTerm[3]=-(L1*(qCurrent[0] + 2*qCurrent[3] - xUp[0] - 2*xUp[3])) - (L3*(qCurrent[0]*qCurrent[0] + qCurrentDerivative[5]*qCurrentDerivative[5] + qCurrentDerivative[6]*qCurrentDerivative[6] + qCurrentDerivative[7]*qCurrentDerivative[7] + qCurrentDerivative[8]*qCurrentDerivative[8] + qCurrentDerivative[9]*qCurrentDerivative[9] - qCurrentDerivative[10]*qCurrentDerivative[10] - qCurrentDerivative[11]*qCurrentDerivative[11] - qCurrentDerivative[12]*qCurrentDerivative[12] - qCurrentDerivative[13]*qCurrentDerivative[13] - qCurrentDerivative[14]*qCurrentDerivative[14] - xUp[0]*xUp[0] + 2*qCurrent[0]*qCurrent[3] - 2*qCurrent[1]*qCurrentDerivative[8] + qCurrentDerivative[5]*(-qCurrent[1] + qCurrentDerivative[8]) - 2*qCurrent[2]*qCurrentDerivative[13] - qCurrentDerivative[10]*(qCurrent[2] + qCurrentDerivative[13]) + 2*qCurrent[3]*xUp[0] - 2*qCurrent[0]*xUp[3] - 2*xUp[0]*xUp[3] - xUp[1]*xUpDerivative[5] - 2*xUp[1]*xUpDerivative[8] - xUp[2]*xUpDerivative[10] - 2*xUp[2]*xUpDerivative[13]))/2.;

            xPlusTerm[4]=-2*L1*(qCurrent[4] - xUp[4]) - (L3*(-2*qCurrent[1]*qCurrentDerivative[9] + 2*qCurrentDerivative[5]*qCurrentDerivative[10] + qCurrentDerivative[8]*qCurrentDerivative[10] + 2*qCurrentDerivative[6]*qCurrentDerivative[11] + 2*qCurrentDerivative[7]*qCurrentDerivative[12] + qCurrentDerivative[5]*qCurrentDerivative[13] + 2*qCurrentDerivative[8]*qCurrentDerivative[13] - 2*qCurrent[2]*qCurrentDerivative[14] + 2*qCurrentDerivative[9]*qCurrentDerivative[14] + 2*qCurrent[0]*(qCurrent[4] - xUp[4]) + 2*xUp[0]*(qCurrent[4] - xUp[4]) - 2*xUp[1]*xUpDerivative[9] - 2*xUp[2]*xUpDerivative[14]))/2.;
            }
        if(d_types[iyd] <= 0) //yMinus
            {
            yMinusTerm[0]=-(L1*(8*qCurrent[0] + 4*qCurrent[3] - 8*yDown[0] - 4*yDown[3]))/4. - (L3*(qCurrent[3]*qCurrent[3] + qCurrentDerivative[0]*qCurrentDerivative[0] + qCurrentDerivative[1]*qCurrentDerivative[1] + qCurrentDerivative[2]*qCurrentDerivative[2] + qCurrentDerivative[3]*qCurrentDerivative[3] + qCurrentDerivative[4]*qCurrentDerivative[4] - qCurrentDerivative[10]*qCurrentDerivative[10] - qCurrentDerivative[11]*qCurrentDerivative[11] - qCurrentDerivative[12]*qCurrentDerivative[12] - qCurrentDerivative[13]*qCurrentDerivative[13] - qCurrentDerivative[14]*qCurrentDerivative[14] - yDown[3]*yDown[3] + 2*qCurrent[0]*qCurrent[3] + qCurrent[1]*qCurrentDerivative[3] + qCurrentDerivative[0]*(2*qCurrent[1] + qCurrentDerivative[3]) + qCurrent[4]*qCurrentDerivative[13] - qCurrentDerivative[10]*(-2*qCurrent[4] + qCurrentDerivative[13]) - 2*qCurrent[3]*yDown[0] + 2*qCurrent[0]*yDown[3] - 2*yDown[0]*yDown[3] + 2*yDown[1]*yDownDerivative[0] + yDown[1]*yDownDerivative[3] + 2*yDown[4]*yDownDerivative[10] + yDown[4]*yDownDerivative[13]))/2.;

            yMinusTerm[1]=-2*L1*(qCurrent[1] - yDown[1]) - (L3*(qCurrentDerivative[3]*(qCurrent[0] + 2*qCurrent[3] - yDown[0] - 2*yDown[3]) + qCurrentDerivative[0]*(2*qCurrent[0] + qCurrent[3] - 2*yDown[0] - yDown[3]) + 2*(qCurrent[1]*qCurrent[3] + qCurrent[2]*qCurrentDerivative[2] + qCurrent[4]*qCurrentDerivative[4] + qCurrent[4]*qCurrentDerivative[11] - qCurrent[3]*yDown[1] - qCurrentDerivative[1]*(-2*qCurrent[1] + yDown[1]) - qCurrentDerivative[2]*yDown[2] + qCurrent[1]*yDown[3] - yDown[1]*yDown[3] - qCurrentDerivative[4]*yDown[4] + yDown[1]*yDownDerivative[1] + yDown[4]*yDownDerivative[11])))/2. - (L2*(2*qCurrent[1] + qCurrentDerivative[0] + qCurrentDerivative[12] - 2*yDown[1] + yDownDerivative[0] + yDownDerivative[12]))/2.;

            yMinusTerm[2]=-2*L1*(qCurrent[2] - yDown[2]) - (L3*(2*qCurrent[1]*qCurrentDerivative[2] + 2*qCurrentDerivative[0]*qCurrentDerivative[10] + qCurrentDerivative[3]*qCurrentDerivative[10] + 2*qCurrentDerivative[1]*qCurrentDerivative[11] + 2*qCurrent[4]*qCurrentDerivative[12] + 2*qCurrentDerivative[2]*qCurrentDerivative[12] + qCurrentDerivative[0]*qCurrentDerivative[13] + 2*qCurrentDerivative[3]*qCurrentDerivative[13] + 2*qCurrentDerivative[4]*qCurrentDerivative[14] + 2*qCurrent[3]*(qCurrent[2] - yDown[2]) + 2*(qCurrent[2] - yDown[2])*yDown[3] + 2*yDown[1]*yDownDerivative[2] + 2*yDown[4]*yDownDerivative[12]))/2.;

            yMinusTerm[3]=-(L1*(4*qCurrent[0] + 8*qCurrent[3] - 4*yDown[0] - 8*yDown[3]))/4. - (L3*(qCurrent[0]*qCurrent[0] + qCurrent[1]*qCurrent[1] + qCurrent[2]*qCurrent[2] + 3*(qCurrent[3]*qCurrent[3]) + qCurrent[4]*qCurrent[4] - qCurrentDerivative[10]*qCurrentDerivative[10] - qCurrentDerivative[11]*qCurrentDerivative[11] - qCurrentDerivative[12]*qCurrentDerivative[12] - qCurrentDerivative[13]*qCurrentDerivative[13] - qCurrentDerivative[14]*qCurrentDerivative[14] + yDown[0]*yDown[0] + yDown[1]*yDown[1] + yDown[2]*yDown[2] - yDown[3]*yDown[3] + yDown[4]*yDown[4] + 2*qCurrent[0]*qCurrent[3] + qCurrent[1]*qCurrentDerivative[0] + 2*qCurrent[1]*qCurrentDerivative[3] + qCurrentDerivative[10]*(qCurrent[4] - qCurrentDerivative[13]) + 2*qCurrent[4]*qCurrentDerivative[13] - 2*qCurrent[0]*yDown[0] - 2*qCurrent[3]*yDown[0] - 2*qCurrent[1]*yDown[1] - 2*qCurrent[2]*yDown[2] - 2*qCurrent[3]*yDown[3] - 2*qCurrent[4]*yDown[4] + yDown[1]*yDownDerivative[0] + 2*yDown[1]*yDownDerivative[3] + yDown[4]*yDownDerivative[10] + 2*yDown[4]*yDownDerivative[13]))/2. - (L2*(2*qCurrent[3] + qCurrentDerivative[1] + qCurrentDerivative[14] - 2*yDown[3] + yDownDerivative[1] + yDownDerivative[14]))/2.;

            yMinusTerm[4]=-2*L1*(qCurrent[4] - yDown[4]) + (L2*(-2*qCurrent[4] - qCurrentDerivative[2] + qCurrentDerivative[10] + qCurrentDerivative[13] + 2*yDown[4] - yDownDerivative[2] + yDownDerivative[10] + yDownDerivative[13]))/2. - (L3*(qCurrentDerivative[13]*(qCurrent[0] + 2*qCurrent[3] - yDown[0] - 2*yDown[3]) + qCurrentDerivative[10]*(2*qCurrent[0] + qCurrent[3] - 2*yDown[0] - yDown[3]) + 2*(qCurrent[3]*qCurrent[4] + qCurrent[1]*qCurrentDerivative[4] + qCurrent[2]*qCurrentDerivative[12] + 2*qCurrent[4]*qCurrentDerivative[14] + qCurrentDerivative[11]*(qCurrent[1] - yDown[1]) - qCurrentDerivative[12]*yDown[2] + qCurrent[4]*yDown[3] - qCurrent[3]*yDown[4] - qCurrentDerivative[14]*yDown[4] - yDown[3]*yDown[4] + yDown[1]*yDownDerivative[4] + yDown[4]*yDownDerivative[14])))/2.;
            }

        if(d_types[iyu] <= 0) //yPlus
            {
            yPlusTerm[0]=-(L1*(2*qCurrent[0] + qCurrent[3] - 2*yUp[0] - yUp[3])) - (L3*(qCurrent[3]*qCurrent[3] + qCurrentDerivative[0]*qCurrentDerivative[0] + qCurrentDerivative[1]*qCurrentDerivative[1] + qCurrentDerivative[2]*qCurrentDerivative[2] + qCurrentDerivative[3]*qCurrentDerivative[3] + qCurrentDerivative[4]*qCurrentDerivative[4] - qCurrentDerivative[10]*qCurrentDerivative[10] - qCurrentDerivative[11]*qCurrentDerivative[11] - qCurrentDerivative[12]*qCurrentDerivative[12] - qCurrentDerivative[13]*qCurrentDerivative[13] - qCurrentDerivative[14]*qCurrentDerivative[14] - yUp[3]*yUp[3] + 2*qCurrent[0]*qCurrent[3] - qCurrent[1]*qCurrentDerivative[3] + qCurrentDerivative[0]*(-2*qCurrent[1] + qCurrentDerivative[3]) - qCurrent[4]*qCurrentDerivative[13] - qCurrentDerivative[10]*(2*qCurrent[4] + qCurrentDerivative[13]) - 2*qCurrent[3]*yUp[0] + 2*qCurrent[0]*yUp[3] - 2*yUp[0]*yUp[3] - 2*yUp[1]*yUpDerivative[0] - yUp[1]*yUpDerivative[3] - 2*yUp[4]*yUpDerivative[10] - yUp[4]*yUpDerivative[13]))/2.;

            yPlusTerm[1]=-2*L1*(qCurrent[1] - yUp[1]) - (L3*(qCurrentDerivative[0]*(-2*qCurrent[0] - qCurrent[3] + 2*yUp[0] + yUp[3]) + qCurrentDerivative[3]*(-qCurrent[0] - 2*qCurrent[3] + yUp[0] + 2*yUp[3]) - 2*(-(qCurrent[1]*qCurrent[3]) + qCurrent[2]*qCurrentDerivative[2] + qCurrent[4]*qCurrentDerivative[4] + qCurrent[4]*qCurrentDerivative[11] + qCurrentDerivative[1]*(2*qCurrent[1] - yUp[1]) + qCurrent[3]*yUp[1] - qCurrentDerivative[2]*yUp[2] - qCurrent[1]*yUp[3] + yUp[1]*yUp[3] - qCurrentDerivative[4]*yUp[4] + yUp[1]*yUpDerivative[1] + yUp[4]*yUpDerivative[11])))/2. + (L2*(-2*qCurrent[1] + qCurrentDerivative[0] + qCurrentDerivative[12] + 2*yUp[1] + yUpDerivative[0] + yUpDerivative[12]))/2.;

            yPlusTerm[2]=-2*L1*(qCurrent[2] - yUp[2]) - (L3*(-2*qCurrent[1]*qCurrentDerivative[2] + 2*qCurrentDerivative[0]*qCurrentDerivative[10] + qCurrentDerivative[3]*qCurrentDerivative[10] + 2*qCurrentDerivative[1]*qCurrentDerivative[11] - 2*qCurrent[4]*qCurrentDerivative[12] + 2*qCurrentDerivative[2]*qCurrentDerivative[12] + qCurrentDerivative[0]*qCurrentDerivative[13] + 2*qCurrentDerivative[3]*qCurrentDerivative[13] + 2*qCurrentDerivative[4]*qCurrentDerivative[14] + 2*qCurrent[3]*(qCurrent[2] - yUp[2]) + 2*(qCurrent[2] - yUp[2])*yUp[3] - 2*yUp[1]*yUpDerivative[2] - 2*yUp[4]*yUpDerivative[12]))/2.;

            yPlusTerm[3]=-(L1*(qCurrent[0] + 2*qCurrent[3] - yUp[0] - 2*yUp[3])) + (L3*(-(qCurrent[0]*qCurrent[0]) - qCurrent[1]*qCurrent[1] - qCurrent[2]*qCurrent[2] - 3*(qCurrent[3]*qCurrent[3]) - qCurrent[4]*qCurrent[4] + qCurrentDerivative[10]*qCurrentDerivative[10] + qCurrentDerivative[11]*qCurrentDerivative[11] + qCurrentDerivative[12]*qCurrentDerivative[12] + qCurrentDerivative[13]*qCurrentDerivative[13] + qCurrentDerivative[14]*qCurrentDerivative[14] - yUp[0]*yUp[0] - yUp[1]*yUp[1] - yUp[2]*yUp[2] + yUp[3]*yUp[3] - yUp[4]*yUp[4] - 2*qCurrent[0]*qCurrent[3] + qCurrent[1]*qCurrentDerivative[0] + 2*qCurrent[1]*qCurrentDerivative[3] + 2*qCurrent[4]*qCurrentDerivative[13] + qCurrentDerivative[10]*(qCurrent[4] + qCurrentDerivative[13]) + 2*qCurrent[0]*yUp[0] + 2*qCurrent[3]*yUp[0] + 2*qCurrent[1]*yUp[1] + 2*qCurrent[2]*yUp[2] + 2*qCurrent[3]*yUp[3] + 2*qCurrent[4]*yUp[4] + yUp[1]*yUpDerivative[0] + 2*yUp[1]*yUpDerivative[3] + yUp[4]*yUpDerivative[10] + 2*yUp[4]*yUpDerivative[13]))/2. + (L2*(-2*qCurrent[3] + qCurrentDerivative[1] + qCurrentDerivative[14] + 2*yUp[3] + yUpDerivative[1] + yUpDerivative[14]))/2.;

            yPlusTerm[4]=-2*L1*(qCurrent[4] - yUp[4]) - (L2*(2*qCurrent[4] - qCurrentDerivative[2] + qCurrentDerivative[10] + qCurrentDerivative[13] - 2*yUp[4] - yUpDerivative[2] + yUpDerivative[10] + yUpDerivative[13]))/2. - (L3*(qCurrentDerivative[10]*(-2*qCurrent[0] - qCurrent[3] + 2*yUp[0] + yUp[3]) + qCurrentDerivative[13]*(-qCurrent[0] - 2*qCurrent[3] + yUp[0] + 2*yUp[3]) - 2*(-(qCurrent[3]*qCurrent[4]) + qCurrent[1]*qCurrentDerivative[4] + qCurrent[2]*qCurrentDerivative[12] + 2*qCurrent[4]*qCurrentDerivative[14] + qCurrentDerivative[11]*(qCurrent[1] - yUp[1]) - qCurrentDerivative[12]*yUp[2] - qCurrent[4]*yUp[3] + qCurrent[3]*yUp[4] - qCurrentDerivative[14]*yUp[4] + yUp[3]*yUp[4] + yUp[1]*yUpDerivative[4] + yUp[4]*yUpDerivative[14])))/2.;
            }

        if(d_types[izd] <= 0) //zMinus
            {
            zMinusTerm[0]=-(L1*(8*qCurrent[0] + 4*qCurrent[3] - 8*zDown[0] - 4*zDown[3]))/4. - (L3*(-3*(qCurrent[0]*qCurrent[0]) - qCurrent[1]*qCurrent[1] - qCurrent[2]*qCurrent[2] - 2*(qCurrent[3]*qCurrent[3]) - qCurrent[4]*qCurrent[4] + qCurrentDerivative[0]*qCurrentDerivative[0] + qCurrentDerivative[1]*qCurrentDerivative[1] + qCurrentDerivative[2]*qCurrentDerivative[2] + qCurrentDerivative[3]*qCurrentDerivative[3] + qCurrentDerivative[4]*qCurrentDerivative[4] + zDown[0]*zDown[0] - zDown[1]*zDown[1] - zDown[2]*zDown[2] - zDown[4]*zDown[4] - 4*qCurrent[0]*qCurrent[3] + qCurrent[2]*qCurrentDerivative[3] + qCurrentDerivative[0]*(2*qCurrent[2] + qCurrentDerivative[3]) + 2*qCurrent[4]*qCurrentDerivative[5] + qCurrent[4]*qCurrentDerivative[8] + 2*qCurrent[0]*zDown[0] + 2*qCurrent[3]*zDown[0] + 2*qCurrent[1]*zDown[1] + 2*qCurrent[2]*zDown[2] + 2*qCurrent[3]*zDown[3] + 2*zDown[0]*zDown[3] + 2*qCurrent[4]*zDown[4] + 2*zDown[2]*zDownDerivative[0] + zDown[2]*zDownDerivative[3] + 2*zDown[4]*zDownDerivative[5] + zDown[4]*zDownDerivative[8]))/2. + (L2*(-2*qCurrent[0] - 2*qCurrent[3] + qCurrentDerivative[2] + qCurrentDerivative[9] + 2*zDown[0] + 2*zDown[3] + zDownDerivative[2] + zDownDerivative[9]))/2.;

            zMinusTerm[1]=-2*L1*(qCurrent[1] - zDown[1]) - (L3*(2*qCurrent[2]*qCurrentDerivative[1] + 2*qCurrentDerivative[0]*qCurrentDerivative[5] + qCurrentDerivative[3]*qCurrentDerivative[5] + 2*qCurrent[4]*qCurrentDerivative[6] + 2*qCurrentDerivative[1]*qCurrentDerivative[6] + 2*qCurrentDerivative[2]*qCurrentDerivative[7] + qCurrentDerivative[0]*qCurrentDerivative[8] + 2*qCurrentDerivative[3]*qCurrentDerivative[8] + 2*qCurrentDerivative[4]*qCurrentDerivative[9] + 2*qCurrent[0]*(-qCurrent[1] + zDown[1]) + 2*qCurrent[3]*(-qCurrent[1] + zDown[1]) + 2*zDown[0]*(-qCurrent[1] + zDown[1]) + 2*(-qCurrent[1] + zDown[1])*zDown[3] + 2*zDown[2]*zDownDerivative[1] + 2*zDown[4]*zDownDerivative[6]))/2.;

            zMinusTerm[2]=-2*L1*(qCurrent[2] - zDown[2]) - (L2*(2*qCurrent[2] + qCurrentDerivative[0] + qCurrentDerivative[6] - 2*zDown[2] + zDownDerivative[0] + zDownDerivative[6]))/2. - (L3*(2*qCurrent[2]*qCurrentDerivative[2] + 2*qCurrent[4]*qCurrentDerivative[7] + 2*qCurrentDerivative[0]*(qCurrent[0] - zDown[0]) + qCurrentDerivative[3]*(qCurrent[0] - zDown[0]) + 2*qCurrentDerivative[1]*(qCurrent[1] - zDown[1]) + 2*qCurrentDerivative[2]*(qCurrent[2] - zDown[2]) + 2*qCurrent[0]*(-qCurrent[2] + zDown[2]) + 2*qCurrent[3]*(-qCurrent[2] + zDown[2]) + 2*zDown[0]*(-qCurrent[2] + zDown[2]) + qCurrentDerivative[0]*(qCurrent[3] - zDown[3]) + 2*qCurrentDerivative[3]*(qCurrent[3] - zDown[3]) + 2*(-qCurrent[2] + zDown[2])*zDown[3] + 2*qCurrentDerivative[4]*(qCurrent[4] - zDown[4]) + 2*zDown[2]*zDownDerivative[2] + 2*zDown[4]*zDownDerivative[7]))/2.;

            zMinusTerm[3]=-(L1*(4*qCurrent[0] + 8*qCurrent[3] - 4*zDown[0] - 8*zDown[3]))/4. - (L3*(-2*(qCurrent[0]*qCurrent[0]) - qCurrent[1]*qCurrent[1] - qCurrent[2]*qCurrent[2] - 3*(qCurrent[3]*qCurrent[3]) - qCurrent[4]*qCurrent[4] + qCurrentDerivative[5]*qCurrentDerivative[5] + qCurrentDerivative[6]*qCurrentDerivative[6] + qCurrentDerivative[7]*qCurrentDerivative[7] + qCurrentDerivative[8]*qCurrentDerivative[8] + qCurrentDerivative[9]*qCurrentDerivative[9] - zDown[1]*zDown[1] - zDown[2]*zDown[2] + zDown[3]*zDown[3] - zDown[4]*zDown[4] - 4*qCurrent[0]*qCurrent[3] + qCurrent[2]*qCurrentDerivative[0] + 2*qCurrent[2]*qCurrentDerivative[3] + 2*qCurrent[4]*qCurrentDerivative[8] + qCurrentDerivative[5]*(qCurrent[4] + qCurrentDerivative[8]) + 2*qCurrent[0]*zDown[0] + 2*qCurrent[1]*zDown[1] + 2*qCurrent[2]*zDown[2] + 2*qCurrent[0]*zDown[3] + 2*qCurrent[3]*zDown[3] + 2*zDown[0]*zDown[3] + 2*qCurrent[4]*zDown[4] + zDown[2]*zDownDerivative[0] + 2*zDown[2]*zDownDerivative[3] + zDown[4]*zDownDerivative[5] + 2*zDown[4]*zDownDerivative[8]))/2. + (L2*(-2*qCurrent[0] - 2*qCurrent[3] + qCurrentDerivative[2] + qCurrentDerivative[9] + 2*zDown[0] + 2*zDown[3] + zDownDerivative[2] + zDownDerivative[9]))/2.;

            zMinusTerm[4]=-2*L1*(qCurrent[4] - zDown[4]) - (L2*(2*qCurrent[4] + qCurrentDerivative[1] + qCurrentDerivative[8] - 2*zDown[4] + zDownDerivative[1] + zDownDerivative[8]))/2. - (L3*(2*qCurrent[2]*qCurrentDerivative[4] + 2*qCurrent[4]*qCurrentDerivative[9] + 2*qCurrentDerivative[5]*(qCurrent[0] - zDown[0]) + qCurrentDerivative[8]*(qCurrent[0] - zDown[0]) + 2*qCurrentDerivative[6]*(qCurrent[1] - zDown[1]) + 2*qCurrentDerivative[7]*(qCurrent[2] - zDown[2]) + qCurrentDerivative[5]*(qCurrent[3] - zDown[3]) + 2*qCurrentDerivative[8]*(qCurrent[3] - zDown[3]) + 2*qCurrentDerivative[9]*(qCurrent[4] - zDown[4]) + 2*qCurrent[0]*(-qCurrent[4] + zDown[4]) + 2*qCurrent[3]*(-qCurrent[4] + zDown[4]) + 2*zDown[0]*(-qCurrent[4] + zDown[4]) + 2*zDown[3]*(-qCurrent[4] + zDown[4]) + 2*zDown[2]*zDownDerivative[4] + 2*zDown[4]*zDownDerivative[9]))/2.;
            }

        if(d_types[izu] <= 0) //zPlus
            {
            zPlusTerm[0]=-(L1*(2*qCurrent[0] + qCurrent[3] - 2*zUp[0] - zUp[3])) + (L3*(3*(qCurrent[0]*qCurrent[0]) + qCurrent[1]*qCurrent[1] + qCurrent[2]*qCurrent[2] + 2*(qCurrent[3]*qCurrent[3]) + qCurrent[4]*qCurrent[4] - qCurrentDerivative[0]*qCurrentDerivative[0] - qCurrentDerivative[1]*qCurrentDerivative[1] - qCurrentDerivative[2]*qCurrentDerivative[2] - qCurrentDerivative[3]*qCurrentDerivative[3] - qCurrentDerivative[4]*qCurrentDerivative[4] - zUp[0]*zUp[0] + zUp[1]*zUp[1] + zUp[2]*zUp[2] + zUp[4]*zUp[4] + 4*qCurrent[0]*qCurrent[3] + qCurrent[2]*qCurrentDerivative[3] - qCurrentDerivative[0]*(-2*qCurrent[2] + qCurrentDerivative[3]) + 2*qCurrent[4]*qCurrentDerivative[5] + qCurrent[4]*qCurrentDerivative[8] - 2*qCurrent[0]*zUp[0] - 2*qCurrent[3]*zUp[0] - 2*qCurrent[1]*zUp[1] - 2*qCurrent[2]*zUp[2] - 2*qCurrent[3]*zUp[3] - 2*zUp[0]*zUp[3] - 2*qCurrent[4]*zUp[4] + 2*zUp[2]*zUpDerivative[0] + zUp[2]*zUpDerivative[3] + 2*zUp[4]*zUpDerivative[5] + zUp[4]*zUpDerivative[8]))/2. - (L2*(2*qCurrent[0] + 2*qCurrent[3] + qCurrentDerivative[2] + qCurrentDerivative[9] - 2*zUp[0] - 2*zUp[3] + zUpDerivative[2] + zUpDerivative[9]))/2.;

            zPlusTerm[1]=-2*L1*(qCurrent[1] - zUp[1]) - (L3*(-2*qCurrent[2]*qCurrentDerivative[1] + 2*qCurrentDerivative[0]*qCurrentDerivative[5] + qCurrentDerivative[3]*qCurrentDerivative[5] - 2*qCurrent[4]*qCurrentDerivative[6] + 2*qCurrentDerivative[1]*qCurrentDerivative[6] + 2*qCurrentDerivative[2]*qCurrentDerivative[7] + qCurrentDerivative[0]*qCurrentDerivative[8] + 2*qCurrentDerivative[3]*qCurrentDerivative[8] + 2*qCurrentDerivative[4]*qCurrentDerivative[9] + 2*qCurrent[0]*(-qCurrent[1] + zUp[1]) + 2*qCurrent[3]*(-qCurrent[1] + zUp[1]) + 2*zUp[0]*(-qCurrent[1] + zUp[1]) + 2*(-qCurrent[1] + zUp[1])*zUp[3] - 2*zUp[2]*zUpDerivative[1] - 2*zUp[4]*zUpDerivative[6]))/2.;

            zPlusTerm[2]=-2*L1*(qCurrent[2] - zUp[2]) + (L2*(-2*qCurrent[2] + qCurrentDerivative[0] + qCurrentDerivative[6] + 2*zUp[2] + zUpDerivative[0] + zUpDerivative[6]))/2. - (L3*(qCurrentDerivative[0]*(-2*qCurrent[0] - qCurrent[3] + 2*zUp[0] + zUp[3]) + qCurrentDerivative[3]*(-qCurrent[0] - 2*qCurrent[3] + zUp[0] + 2*zUp[3]) - 2*(qCurrent[0]*qCurrent[2] + qCurrent[2]*qCurrent[3] + qCurrent[4]*qCurrentDerivative[4] + qCurrent[4]*qCurrentDerivative[7] + qCurrent[2]*zUp[0] + qCurrentDerivative[1]*(qCurrent[1] - zUp[1]) + qCurrentDerivative[2]*(2*qCurrent[2] - zUp[2]) - qCurrent[0]*zUp[2] - qCurrent[3]*zUp[2] - zUp[0]*zUp[2] + qCurrent[2]*zUp[3] - zUp[2]*zUp[3] - qCurrentDerivative[4]*zUp[4] + zUp[2]*zUpDerivative[2] + zUp[4]*zUpDerivative[7])))/2.;

            zPlusTerm[3]=-(L1*(qCurrent[0] + 2*qCurrent[3] - zUp[0] - 2*zUp[3])) + (L3*(2*(qCurrent[0]*qCurrent[0]) + qCurrent[1]*qCurrent[1] + qCurrent[2]*qCurrent[2] + 3*(qCurrent[3]*qCurrent[3]) + qCurrent[4]*qCurrent[4] - qCurrentDerivative[5]*qCurrentDerivative[5] - qCurrentDerivative[6]*qCurrentDerivative[6] - qCurrentDerivative[7]*qCurrentDerivative[7] - qCurrentDerivative[8]*qCurrentDerivative[8] - qCurrentDerivative[9]*qCurrentDerivative[9] + zUp[1]*zUp[1] + zUp[2]*zUp[2] - zUp[3]*zUp[3] + zUp[4]*zUp[4] + 4*qCurrent[0]*qCurrent[3] + qCurrent[2]*qCurrentDerivative[0] + 2*qCurrent[2]*qCurrentDerivative[3] + qCurrentDerivative[5]*(qCurrent[4] - qCurrentDerivative[8]) + 2*qCurrent[4]*qCurrentDerivative[8] - 2*qCurrent[0]*zUp[0] - 2*qCurrent[1]*zUp[1] - 2*qCurrent[2]*zUp[2] - 2*qCurrent[0]*zUp[3] - 2*qCurrent[3]*zUp[3] - 2*zUp[0]*zUp[3] - 2*qCurrent[4]*zUp[4] + zUp[2]*zUpDerivative[0] + 2*zUp[2]*zUpDerivative[3] + zUp[4]*zUpDerivative[5] + 2*zUp[4]*zUpDerivative[8]))/2. - (L2*(2*qCurrent[0] + 2*qCurrent[3] + qCurrentDerivative[2] + qCurrentDerivative[9] - 2*zUp[0] - 2*zUp[3] + zUpDerivative[2] + zUpDerivative[9]))/2.;

            zPlusTerm[4]=-2*L1*(qCurrent[4] - zUp[4]) + (L2*(-2*qCurrent[4] + qCurrentDerivative[1] + qCurrentDerivative[8] + 2*zUp[4] + zUpDerivative[1] + zUpDerivative[8]))/2. - (L3*(qCurrentDerivative[5]*(-2*qCurrent[0] - qCurrent[3] + 2*zUp[0] + zUp[3]) + qCurrentDerivative[8]*(-qCurrent[0] - 2*qCurrent[3] + zUp[0] + 2*zUp[3]) - 2*(qCurrent[0]*qCurrent[4] + qCurrent[3]*qCurrent[4] + qCurrent[2]*qCurrentDerivative[4] + 2*qCurrent[4]*qCurrentDerivative[9] + qCurrent[4]*zUp[0] + qCurrentDerivative[6]*(qCurrent[1] - zUp[1]) + qCurrentDerivative[7]*(qCurrent[2] - zUp[2]) + qCurrent[4]*zUp[3] - qCurrent[0]*zUp[4] - qCurrent[3]*zUp[4] - qCurrentDerivative[9]*zUp[4] - zUp[0]*zUp[4] - zUp[3]*zUp[4] + zUp[2]*zUpDerivative[4] + zUp[4]*zUpDerivative[9])))/2.;
            }

        force += xMinusTerm+xPlusTerm+yMinusTerm+yPlusTerm+zMinusTerm+zPlusTerm;

        };
    if(zeroForce)
        d_force[idx] = force;
    else
        d_force[idx] += force;
    }

__global__ void gpu_qTensor_uniformFieldForcekernel(dVec *d_force,
                                                    int *d_types,
                                                    int N,
                                                    scalar3 field,
                                                    scalar anisotropicSusceptibility,
                                                    scalar vacuumPermeability,
                                                    bool zeroForce)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    if(d_types[idx]>0)
        return;
    scalar fieldProduct = anisotropicSusceptibility*vacuumPermeability;
    dVec fieldForce(0.);
    fieldForce[0] = -0.5*fieldProduct*(field.x*field.x-field.z*field.z);
    fieldForce[1] = -fieldProduct*field.x*field.y;
    fieldForce[2] = -fieldProduct*field.x*field.z;
    fieldForce[3] = -0.5*fieldProduct*(field.y*field.y-field.z*field.z);
    fieldForce[4] = -fieldProduct*field.y*field.z;
    if(zeroForce)
        d_force[idx] = fieldForce;
    else
        d_force[idx] -= fieldForce;
    }

__global__ void gpu_qTensor_l24Force_kernel(dVec *d_force,
                                int *d_types,
                                cubicLatticeDerivativeVector *d_derivatives,
                                Index3D latticeIndex,
                                int N,
                                scalar L24,
                                bool zeroForce)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    int3 target = latticeIndex.inverseIndex(idx);
    int3 latticeSizes = latticeIndex.getSizes();
    dVec force(0.0);

    if(d_types[idx] <= 0) //no force on sites that are part of boundaries
        {
        //get neighbor indices and data
        int ixd, ixu,iyd,iyu,izd,izu;
        gpu_get_six_neighbors(target,ixd, ixu,iyd,iyu,izd,izu,latticeIndex,latticeSizes);
        cubicLatticeDerivativeVector qCurrentDerivative = d_derivatives[idx];
        cubicLatticeDerivativeVector xDownDerivative = d_derivatives[ixd];
        cubicLatticeDerivativeVector xUpDerivative = d_derivatives[ixu];
        cubicLatticeDerivativeVector yDownDerivative = d_derivatives[iyd];
        cubicLatticeDerivativeVector yUpDerivative = d_derivatives[iyu];
        cubicLatticeDerivativeVector zDownDerivative = d_derivatives[izd];
        cubicLatticeDerivativeVector zUpDerivative = d_derivatives[izu];

        dVec xMinusTerm(0.0);
        dVec xPlusTerm(0.0);
        dVec yMinusTerm(0.0);
        dVec yPlusTerm(0.0);
        dVec zMinusTerm(0.0);
        dVec zPlusTerm(0.0);
        if(d_types[ixd] <= 0) //xMinus
            {
            xMinusTerm[0]=(3*L24*(qCurrentDerivative[6] + xDownDerivative[6] + 2*(qCurrentDerivative[12] + xDownDerivative[12])))/2.;

            xMinusTerm[1]=(-3*L24*(qCurrentDerivative[5] - qCurrentDerivative[8] - qCurrentDerivative[14] + xDownDerivative[5] - xDownDerivative[8] - xDownDerivative[14]))/2.;

            xMinusTerm[2]=(-3*L24*(-qCurrentDerivative[9] + 2*qCurrentDerivative[10] + qCurrentDerivative[13] - xDownDerivative[9] + 2*xDownDerivative[10] + xDownDerivative[13]))/2.;

            xMinusTerm[3]=(-3*L24*(qCurrentDerivative[6] - qCurrentDerivative[12] + xDownDerivative[6] - xDownDerivative[12]))/2.;

            xMinusTerm[4]=(-3*L24*(qCurrentDerivative[7] + qCurrentDerivative[11] + xDownDerivative[7] + xDownDerivative[11]))/2.;
            }
        if(d_types[ixu] <= 0) //xPlus
            {
            xPlusTerm[0]=(-3*L24*(qCurrentDerivative[6] + xUpDerivative[6] + 2*(qCurrentDerivative[12] + xUpDerivative[12])))/2.;

            xPlusTerm[1]=(-3*L24*(-qCurrentDerivative[5] + qCurrentDerivative[8] + qCurrentDerivative[14] - xUpDerivative[5] + xUpDerivative[8] + xUpDerivative[14]))/2.;

            xPlusTerm[2]=(3*L24*(-qCurrentDerivative[9] + 2*qCurrentDerivative[10] + qCurrentDerivative[13] - xUpDerivative[9] + 2*xUpDerivative[10] + xUpDerivative[13]))/2.;

            xPlusTerm[3]=(3*L24*(qCurrentDerivative[6] - qCurrentDerivative[12] + xUpDerivative[6] - xUpDerivative[12]))/2.;

            xPlusTerm[4]=(3*L24*(qCurrentDerivative[7] + qCurrentDerivative[11] + xUpDerivative[7] + xUpDerivative[11]))/2.;
            }

        if(d_types[iyd] <= 0) //yMinus
            {
            yMinusTerm[0]=(-3*L24*(qCurrentDerivative[1] - qCurrentDerivative[14] + yDownDerivative[1] - yDownDerivative[14]))/2.;

            yMinusTerm[1]=(3*L24*(qCurrentDerivative[0] - qCurrentDerivative[3] + qCurrentDerivative[12] + yDownDerivative[0] - yDownDerivative[3] + yDownDerivative[12]))/2.;

            yMinusTerm[2]=(-3*L24*(qCurrentDerivative[4] + qCurrentDerivative[11] + yDownDerivative[4] + yDownDerivative[11]))/2.;

            yMinusTerm[3]=(3*L24*(qCurrentDerivative[1] + yDownDerivative[1] + 2*(qCurrentDerivative[14] + yDownDerivative[14])))/2.;

            yMinusTerm[4]=(-3*L24*(-qCurrentDerivative[2] + qCurrentDerivative[10] + 2*qCurrentDerivative[13] - yDownDerivative[2] + yDownDerivative[10] + 2*yDownDerivative[13]))/2.;
            }

        if(d_types[iyu] <= 0) //yPlus
            {
            yPlusTerm[0]=(3*L24*(qCurrentDerivative[1] - qCurrentDerivative[14] + yUpDerivative[1] - yUpDerivative[14]))/2.;

            yPlusTerm[1]=(-3*L24*(qCurrentDerivative[0] - qCurrentDerivative[3] + qCurrentDerivative[12] + yUpDerivative[0] - yUpDerivative[3] + yUpDerivative[12]))/2.;

            yPlusTerm[2]=(3*L24*(qCurrentDerivative[4] + qCurrentDerivative[11] + yUpDerivative[4] + yUpDerivative[11]))/2.;

            yPlusTerm[3]=(-3*L24*(qCurrentDerivative[1] + yUpDerivative[1] + 2*(qCurrentDerivative[14] + yUpDerivative[14])))/2.;

            yPlusTerm[4]=(3*L24*(-qCurrentDerivative[2] + qCurrentDerivative[10] + 2*qCurrentDerivative[13] - yUpDerivative[2] + yUpDerivative[10] + 2*yUpDerivative[13]))/2.;
            }

        if(d_types[izd] <= 0) //zMinus
            {
            zMinusTerm[0]=(-3*L24*(2*qCurrentDerivative[2] + qCurrentDerivative[9] + 2*zDownDerivative[2] + zDownDerivative[9]))/2.;

            zMinusTerm[1]=(-3*L24*(qCurrentDerivative[4] + qCurrentDerivative[7] + zDownDerivative[4] + zDownDerivative[7]))/2.;

            zMinusTerm[2]=(3*L24*(2*qCurrentDerivative[0] + qCurrentDerivative[3] + qCurrentDerivative[6] + 2*zDownDerivative[0] + zDownDerivative[3] + zDownDerivative[6]))/2.;

            zMinusTerm[3]=(-3*L24*(qCurrentDerivative[2] + zDownDerivative[2] + 2*(qCurrentDerivative[9] + zDownDerivative[9])))/2.;

            zMinusTerm[4]=(3*L24*(qCurrentDerivative[1] + qCurrentDerivative[5] + 2*qCurrentDerivative[8] + zDownDerivative[1] + zDownDerivative[5] + 2*zDownDerivative[8]))/2.;
            }

        if(d_types[izu] <= 0) //zPlus
            {
            zPlusTerm[0]=(3*L24*(2*qCurrentDerivative[2] + qCurrentDerivative[9] + 2*zUpDerivative[2] + zUpDerivative[9]))/2.;

            zPlusTerm[1]=(3*L24*(qCurrentDerivative[4] + qCurrentDerivative[7] + zUpDerivative[4] + zUpDerivative[7]))/2.;

            zPlusTerm[2]=(-3*L24*(2*qCurrentDerivative[0] + qCurrentDerivative[3] + qCurrentDerivative[6] + 2*zUpDerivative[0] + zUpDerivative[3] + zUpDerivative[6]))/2.;

            zPlusTerm[3]=(3*L24*(qCurrentDerivative[2] + zUpDerivative[2] + 2*(qCurrentDerivative[9] + zUpDerivative[9])))/2.;

            zPlusTerm[4]=(-3*L24*(qCurrentDerivative[1] + qCurrentDerivative[5] + 2*qCurrentDerivative[8] + zUpDerivative[1] + zUpDerivative[5] + 2*zUpDerivative[8]))/2.;
            }

        force += xMinusTerm+xPlusTerm+yMinusTerm+yPlusTerm+zMinusTerm+zPlusTerm;
        };
    if(zeroForce)
        d_force[idx] = force;
    else
        d_force[idx] += force;
    };

bool gpu_qTensor_computeBoundaryForcesGPU(dVec *d_force,
                                dVec *d_spins,
                                int *d_types,
                                boundaryObject *d_bounds,
                                Index3D latticeIndex,
                                int N,
                                bool zeroForce,
                                int maxBlockSize)
    {
    unsigned int block_size = maxBlockSize;
    unsigned int nblocks = N/block_size+1;
    gpu_qTensor_computeBoundaryForcesGPU_kernel<<<nblocks,block_size>>>(d_force,d_spins,d_types,d_bounds,latticeIndex,
                                                             N,zeroForce);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

bool gpu_qTensor_firstDerivatives(cubicLatticeDerivativeVector *d_derivatives,
                          dVec *d_spins,
                          int *d_types,
                          Index3D latticeIndex,
                          int N,
                          int maxBlockSize)
    {
    unsigned int block_size = maxBlockSize;
    unsigned int nblocks = N/block_size+1;
    gpu_qTensor_firstDerivatives_kernel<<<nblocks,block_size>>>(d_derivatives,d_spins,d_types,latticeIndex,N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

bool gpu_qTensor_oneConstantForce(dVec *d_force,
                                dVec *d_spins,
                                int *d_types,
                                int *d_latticeNeighbors,
                                Index3D latticeIndex,
                                Index2D neighborIndex,
                                scalar A,scalar B,scalar C,scalar L,
                                int N,
                                bool zeroForce,
                                int maxBlockSize)
    {
    unsigned int block_size = maxBlockSize;
    unsigned int nblocks = N/block_size+1;
    scalar a = 0.5*A;
    scalar b = B/3.0;
    scalar c = 0.25*C;
    scalar l = L;
    gpu_qTensor_oneConstantForce_kernel<<<nblocks,block_size>>>(d_force,d_spins,d_types,d_latticeNeighbors,
                                                                latticeIndex,neighborIndex,
                                                                a,b,c,l,N,zeroForce);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

bool gpu_qTensor_twoConstantForce(dVec *d_force,
                                dVec *d_spins,
                                int *d_types,
                                cubicLatticeDerivativeVector *d_derivatives,
                                Index3D latticeIndex,
                                scalar A,scalar B,scalar C,scalar L1,scalar L2, scalar q0,
                                int N,
                                bool zeroForce,
                                int maxBlockSize)
    {
    unsigned int block_size = maxBlockSize;
    unsigned int nblocks = N/block_size+1;
    scalar a = 0.5*A;
    scalar b = B/3.0;
    scalar c = 0.25*C;
    gpu_qTensor_twoConstantForce_kernel<<<nblocks,block_size>>>(d_force,d_spins,d_types,d_derivatives,latticeIndex,
                                                             a,b,c,L1,L2,q0,N,zeroForce);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

bool gpu_qTensor_threeConstantForce(dVec *d_force,
                                dVec *d_spins,
                                int *d_types,
                                cubicLatticeDerivativeVector *d_derivatives,
                                Index3D latticeIndex,
                                scalar A,scalar B,scalar C,scalar L1,scalar L2, scalar L3,
                                int N,
                                bool zeroForce,
                                int maxBlockSize)
    {
    unsigned int block_size = maxBlockSize;
    unsigned int nblocks = N/block_size+1;
    scalar a = 0.5*A;
    scalar b = B/3.0;
    scalar c = 0.25*C;
    gpu_qTensor_threeConstantForce_kernel<<<nblocks,block_size>>>(d_force,d_spins,d_types,d_derivatives,latticeIndex,
                                                             a,b,c,L1,L2,L3,N,zeroForce);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

bool gpu_qTensor_computeL24ForcesGPU(dVec *d_force,
                                    int *d_types,
                                    cubicLatticeDerivativeVector *d_derivatives,
                                    Index3D latticeIndex,
                                    int N, scalar L24,
                                    bool zeroOutForce,
                                    int maxBlockSize)
    {
    unsigned int block_size = maxBlockSize;
    unsigned int nblocks = N/block_size+1;
    gpu_qTensor_l24Force_kernel<<<nblocks,block_size>>>(d_force,d_types,d_derivatives,latticeIndex,
                                                                 N,L24,zeroOutForce);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };
bool gpu_qTensor_computeUniformFieldForcesGPU(dVec * d_force,
                                       int *d_types,
                                       int N,
                                       scalar3 field,
                                       scalar anisotropicSusceptibility,
                                       scalar vacuumPermeability,
                                       bool zeroOutForce,
                                       int maxBlockSize)
    {
    unsigned int block_size = maxBlockSize;
    unsigned int nblocks = N/block_size+1;
    gpu_qTensor_uniformFieldForcekernel<<<nblocks,block_size>>>(d_force,d_types,N,field,anisotropicSusceptibility,
                                                                vacuumPermeability, zeroOutForce);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };;

/** @} */ //end of group declaration
