#include "hip/hip_runtime.h"
#include "utilities.cuh"
#include "functions.h"

/*! \file utilities.cu
  defines kernel callers and kernels for some simple GPU array calculations

 \addtogroup utilityKernels
 @{
 */

/*!
add the first N elements of array and put it in output[helperIdx]
*/
__global__ void gpu_serial_reduction_kernel(scalar *array, scalar *output, int helperIdx,int N)
    {
    scalar ans = 0.0;
    for (int i = 0; i < N; ++i)
        ans += array[i];
    output[helperIdx] = ans;
    return;
    };

/*!
add the first N elements of array and put it in output[helperIdx]...use shared memory a bit
*/
__global__ void gpu_serial_reduction_kernel2(scalar *array, scalar *output, int helperIdx,int N)
    {
    int tidx = threadIdx.x;
    extern __shared__ scalar partialSum[];

    partialSum[tidx] = 0.0;
    __syncthreads();
    int max = N/ blockDim.x+1;
    for (int i = 0; i < max;++i)
        {
        int pos =  blockDim.x *i+tidx;
        if(pos > N) continue;
        partialSum[tidx] += array[pos];
        }
    __syncthreads();
    if(tidx ==0)
        {
        scalar ans =0.0;
        for (int i = 0; i <  blockDim.x; ++i)
            ans += partialSum[i];
        output[helperIdx] = ans;
        }

    return;
    };

/*!
perform a block reduction, storing the partial sums of input into output
*/
__global__ void gpu_parallel_block_reduction_kernel(scalar *input, scalar *output,int N)
    {
    extern __shared__ scalar sharedArray[];

    unsigned int tidx = threadIdx.x;
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    //load into shared memory and synchronize
    if(i < N)
        sharedArray[tidx] = input[i];
    else
        sharedArray[tidx] = 0.0;
    __syncthreads();

    //reduce
    for (int s = blockDim.x/2; s>0; s>>=1)
        {
        if (tidx < s)
            sharedArray[tidx] += sharedArray[tidx+s];
        __syncthreads();
        };
    //write to the correct block of the output array
    if (tidx==0)
        output[blockIdx.x] = sharedArray[0];
    };

/*!
a slight optimization of the previous block reduction, c.f. M. Harris presentation
*/
__global__ void gpu_parallel_block_reduction2_kernel(scalar *input, scalar *output,int N)
    {
    extern __shared__ scalar sharedArray[];
    unsigned int tidx = threadIdx.x;
    unsigned int i = 2*blockDim.x * blockIdx.x + threadIdx.x;

    scalar sum;
    //load into shared memory and synchronize
    if(i < N)
        sum = input[i];
    else
        sum = 0.0;
    if(i + blockDim.x < N)
        sum += input[i+blockDim.x];
    sharedArray[tidx] = sum;
    __syncthreads();

    //reduce
    for (int s = blockDim.x/2; s>0; s>>=1)
        {
        if (tidx < s)
            sharedArray[tidx] = sum = sum+sharedArray[tidx+s];
        __syncthreads();
        };
    //write to the correct block of the output array
    if (tidx==0)
        output[blockIdx.x] = sum;
    };

/*!
  multiple loads and loop unrolling...
a slight optimization of the previous block reduction, c.f. M. Harris presentation
*/
__global__ void gpu_parallel_block_reduction3_kernel(scalar *input, scalar *output,int N)
    {
    extern __shared__ scalar sharedArray[];
    unsigned int tidx = threadIdx.x;
    unsigned int i = 2*blockDim.x * blockIdx.x + threadIdx.x;

    if(i+blockDim.x < N)
        sharedArray[tidx] = input[i]+input[i+blockDim.x];
    else if(i < N)
        sharedArray[tidx] = input[i];
    else
        sharedArray[tidx] = 0.0;
    __syncthreads();

    //reduce
    for (int stride = blockDim.x/2;stride >32; stride >>=1)
        {
        if(tidx<stride)
            sharedArray[tidx] += sharedArray[tidx+stride];
        __syncthreads();
        }
    if(tidx < 32)
        {
        sharedArray[tidx] += sharedArray[tidx+32];
        sharedArray[tidx] += sharedArray[tidx+16];
        sharedArray[tidx] += sharedArray[tidx+8];
        sharedArray[tidx] += sharedArray[tidx+4];
        sharedArray[tidx] += sharedArray[tidx+2];
        sharedArray[tidx] += sharedArray[tidx+1];
        }
    //write to the correct block of the output array
    if (tidx==0)
        output[blockIdx.x] = sharedArray[0];
    };

/*!
Store the dot product of two dVecs in a scalar vec
*/
__global__ void gpu_vec_dot_product_kernel(dVec *input1, dVec *input2, scalar *output,int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    output[idx] = dot(input1[idx],input2[idx]);
    return;
    };


/*!
This kernel basically performs the operation of the "reduction2" kernel, but the shared memory gets
dot products...BROKEN
*/
__global__ void gpu_dVec_dot_products_kernel(dVec *input1, dVec *input2, scalar *output,int N)
    {
    extern __shared__ scalar sharedArray[];
    unsigned int tidx = threadIdx.x;
    unsigned int i = 2*blockDim.x * blockIdx.x + threadIdx.x;

    scalar tempSum;
    if(i < N)
        tempSum = dot(input1[i],input2[i]);
    else
        tempSum = 0.0;
    sharedArray[tidx] = 0.0;
    __syncthreads();

    //reduce
    for (int s = blockDim.x/2;s>0; s>>=1)
        {
        if (tidx <s)
            sharedArray[tidx] = tempSum = tempSum+sharedArray[tidx+s];
        __syncthreads();
        };
    //write to the correct block of the output array
    if(tidx==0)
        output[blockIdx.x] = tempSum;
    };

/*!
This kernel basically performs the operation of the "reduction2" kernel, but the shared memory gets dot products
*/
__global__ void gpu_unrolled_dVec_dot_products_kernel(dVec *input1, dVec *input2, scalar *output,int N)
    {
    extern __shared__ scalar sharedArray[];
    unsigned int tidx = threadIdx.x;
    unsigned int i = 2*blockDim.x * blockIdx.x + threadIdx.x;

    int p1 = i / DIMENSION;
    int d1 = i % DIMENSION;
    int p2 = (i+blockDim.x) / DIMENSION;
    int d2 = (i+blockDim.x) % DIMENSION;

    if(i+blockDim.x < N)
        sharedArray[tidx] = input1[p1][d1]*input2[p1][d1] + input1[p2][d2]*input2[p2][d2];
    else if(i < N)
        sharedArray[tidx] = input1[p1][d1]*input2[p1][d1];
    else
        sharedArray[tidx] = 0.0;
    __syncthreads();

    //reduce
    for (int stride = blockDim.x/2;stride >32; stride >>=1)
        {
        if(tidx<stride)
            sharedArray[tidx] += sharedArray[tidx+stride];
        __syncthreads();
        }
    if(tidx < 32)
        {
        sharedArray[tidx] += sharedArray[tidx+32];
        sharedArray[tidx] += sharedArray[tidx+16];
        sharedArray[tidx] += sharedArray[tidx+8];
        sharedArray[tidx] += sharedArray[tidx+4];
        sharedArray[tidx] += sharedArray[tidx+2];
        sharedArray[tidx] += sharedArray[tidx+1];
        }
    //write to the correct block of the output array
    if (tidx==0)
        output[blockIdx.x] = sharedArray[0];
    };

/*!
take a vector of dVecs, a vector of scalars, a factor, and return a vector where
every entry is
factor*scalar[i]*(dVec[i])^2
*/
__global__ void gpu_scalar_times_dVec_squared_kernel(dVec *d_vec1, scalar *d_scalars, scalar factor, scalar *d_ans, int n)
    {
    // read in the index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= n)
        return;
    d_ans[idx] = factor * d_scalars[idx]*dot(d_vec1[idx],d_vec1[idx]);
    };
/*!
take two vectors of dVecs and return a vector of scalars, where each entry is vec1[i].vec2[i]
*/
__global__ void gpu_dot_dVec_vectors_kernel(dVec *d_vec1, dVec *d_vec2, scalar *d_ans, int n)
    {
    // read in the index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= n)
        return;
    d_ans[idx] = dot(d_vec1[idx],d_vec2[idx]);
    };
/*!
  multiply every element of an array of dVecs by the same scalar
  */
__global__ void gpu_dVec_times_scalar_kernel(dVec *d_vec1,scalar factor, int n)
    {
    // read in the index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= n)
        return;
    d_vec1[idx] = factor*d_vec1[idx];
    };
/*!
  multiply every element of an array of dVecs by the same scalar
  */
__global__ void gpu_dVec_times_scalar_kernel(dVec *d_vec1,scalar factor, dVec *d_ans,int n)
    {
    // read in the index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= n)
        return;
    d_ans[idx] = factor*d_vec1[idx];
    };


__global__ void gpu_dVec_plusEqual_dVec_kernel(dVec *d_vec1,dVec *d_vec2,scalar factor,int n)
    {
    // read in the index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= n)
        return;
    int pIdx = idx / DIMENSION;
    int dIdx = idx % DIMENSION;

    d_vec1[pIdx][dIdx] += factor*d_vec2[pIdx][dIdx];
    };

/////
//Kernel callers
///

bool gpu_dVec_plusEqual_dVec(dVec *d_vec1,
                              dVec *d_vec2,
                              scalar factor,
                              int N,
                              int maxBlockSize)
    {
    unsigned int block_size = maxBlockSize;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = (DIMENSION*N)/block_size + 1;
    gpu_dVec_plusEqual_dVec_kernel<<<nblocks,block_size>>>(d_vec1,d_vec2,factor,DIMENSION*N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };
/*!
\param d_vec1 dVec input array
\param factor scalar multiplication factor
\param N      the length of the arrays
\post d_vec1 *= factor for every element
 */
bool gpu_dVec_times_scalar(dVec *d_vec1, scalar factor, int N)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    gpu_dVec_times_scalar_kernel<<<nblocks,block_size>>>(
                                                d_vec1,
                                                factor,
                                                N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

bool gpu_dVec_times_scalar(dVec *d_vec1, scalar factor, dVec *d_ans,int N)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    gpu_dVec_times_scalar_kernel<<<nblocks,block_size>>>(
                                                d_vec1,
                                                factor,
                                                d_ans,
                                                N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

bool gpu_scalar_times_dVec_squared(dVec *d_vec1, scalar *d_scalars, scalar factor, scalar *d_ans, int N)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    gpu_scalar_times_dVec_squared_kernel<<<nblocks,block_size>>>(
                                                d_vec1,
                                                d_scalars,
                                                factor,
                                                d_ans,
                                                N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/*!
\param d_vec1 dVec input array
\param d_vec2 dVec input array
\param d_ans  scalar output array... d_ans[idx] = d_vec1[idx].d_vec2[idx]
\param N      the length of the arrays
\post d_ans = d_vec1.d_vec2
*/
bool gpu_dot_dVec_vectors(dVec *d_vec1, dVec *d_vec2, scalar *d_ans, int N)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;
    gpu_dot_dVec_vectors_kernel<<<nblocks,block_size>>>(
                                                d_vec1,
                                                d_vec2,
                                                d_ans,
                                                N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/*!
takes the dot product of every element of the two input arrays and performs a reduction on the sum
\param input1 vector 1...wow!
\param input2 vector 2...wow!
\param intermediate an array that input is dot producted to
\param intermediate2 an array that input is block-reduced to
\param output the intermediate array will be sum reduced and stored in one of the components of output
\param helperIdx the location in output to store the answer
\param N the size of the input and  intermediate arrays
\param block_size the...block size. doxygen is annoying sometimes
*/
bool gpu_dVec_dot_products(dVec *input1,dVec *input2, scalar *intermediate, scalar *intermediate2,scalar *output, int helperIdx, int N,int block_size)
    {
    //int problemSize = DIMENSION*N;
    //unsigned int nblocks  = problemSize/block_size + 1;
    unsigned int nblocks  = N/block_size + 1;

    //first dot the vectors together
    gpu_vec_dot_product_kernel<<<nblocks,block_size>>>(input1,input2,intermediate,N);
    HANDLE_ERROR(hipGetLastError());

    //then call the parallel reduction routine to sum up the answer
    gpu_parallel_reduction(intermediate,intermediate2,output,helperIdx,N,block_size);
    //gpu_serial_reduction(intermediate,output,helperIdx,N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    /*
    HANDLE_ERROR(hipGetLastError());
    //first do a block reduction of input
    unsigned int smem = block_size*sizeof(scalar);
    //Do a block reduction of the input array
    //gpu_unrolled_dVec_dot_products_kernel<<<nblocks,block_size,smem>>>(input1,input2,intermediate, problemSize);
    gpu_dVec_dot_products_kernel<<<nblocks,block_size,smem>>>(input1,input2,intermediate, N);
    HANDLE_ERROR(hipGetLastError());

    //sum reduce the temporary array, saving the result in the right slot of the output array
    int nb=1024;
    if(nblocks < nb) nb = 1;
    gpu_serial_reduction_kernel2<<<1,nb,nb*sizeof(scalar)>>>(intermediate,output,helperIdx,nblocks+1);
    HANDLE_ERROR(hipGetLastError());
    */
    }

/*
A stub of a function...eventually replace with off-the-shelf solution?
*/
bool gpu_dVec_dot_products(dVec *input1,dVec *input2, scalar *output, int helperIdx, int N)
    {
    //scalar init = 0.0;
    //dVecDotProduct mult_op;
    //thrust::plus<scalar> add_op;
    //thrust::device_ptr<scalar> ptrAns = thrust::device_pointer_cast(output);
    //thrust::device_ptr<dVec> ptr1 = thrust::device_pointer_cast(input1);
    //thrust::device_ptr<dVec> ptr2 = thrust::device_pointer_cast(input2);
    //output[helperIdx] = thrust::inner_product(thrust::device,ptr1,ptr1+N,ptr2,init,add_op,mult_op);
    //output[helperIdx] = thrust::inner_product(thrust::device,input1,input1+N,input2,init,add_op,mult_op);
    //ptrAns[helperIdx] = thrust::inner_product(thrust::device,input1,input1+N,input2,init,add_op,mult_op);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/*!
a two-step parallel reduction algorithm that first does a partial sum reduction of input into the
intermediate array, then launches a second kernel to sum reduce intermediate into output[helperIdx]
\param input the input array to sum
\param intermediate an array that input is block-reduced to
\param output the intermediate array will be sum reduced and stored in one of the components of output
\param helperIdx the location in output to store the answer
\param N the size of the input and  intermediate arrays
\param block_size the...block size. doxygen is annoying sometimes
*/
bool gpu_parallel_reduction(scalar *input, scalar *intermediate, scalar *output, int helperIdx, int N,int block_size)
    {
    unsigned int nblocks  = N/block_size + 1;
    //first do a block reduction of input
    unsigned int smem = block_size*sizeof(scalar);

    //Do a block reduction of the input array
    gpu_parallel_block_reduction2_kernel<<<nblocks,block_size,smem>>>(input,intermediate, N);
    HANDLE_ERROR(hipGetLastError());

    //sum reduce the temporary array, saving the result in the right slot of the output array
    int nb=1024;
    if(nblocks < nb) nb = 1;
    gpu_serial_reduction_kernel2<<<1,nb,nb*sizeof(scalar)>>>(intermediate,output,helperIdx,nblocks+1);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/*!
This serial reduction routine should probably never be called. It provides an interface to the
gpu_serial_reduction_kernel above that may be useful for testing
  */
bool gpu_serial_reduction(scalar *array, scalar *output, int helperIdx, int N)
    {
    gpu_serial_reduction_kernel<<<1,1>>>(array,output,helperIdx,N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/*!
  A function of convenience... set an array on the device
  */
template <typename T>
__global__ void gpu_set_array_kernel(T *arr,T value, int N)
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    arr[idx] = value;
    return;
    };

template<typename T>
bool gpu_set_array(T *array, T value, int N,int maxBlockSize)
    {
    unsigned int block_size = maxBlockSize;
    if (N < 128) block_size = 16;
    unsigned int nblocks  = N/block_size + 1;
    gpu_set_array_kernel<<<nblocks, block_size>>>(array,value,N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

scalar host_dVec_dot_products(dVec *input1,dVec *input2,int N)
    {
    scalar ans = 0.0;
    for (int ii = 0; ii < N; ++ii)
        for (int dd = 0; dd < DIMENSION; ++dd)
            ans +=input1[ii][dd]*input2[ii][dd];
    return ans;
    }

void host_dVec_plusEqual_dVec(dVec *d_vec1,dVec *d_vec2,scalar factor,int N)
    {
    for (int ii = 0; ii < N; ++ii)
        d_vec1[ii] = d_vec1[ii] + factor*d_vec2[ii];
    }

void host_dVec_times_scalar(dVec *d_vec1, scalar factor, dVec *d_ans, int N)
    {
    for(int ii = 0; ii < N; ++ii)
        d_ans[ii] = factor*d_vec1[ii];
    }
//explicit template instantiations

template bool gpu_set_array<int>(int *,int, int, int);
template bool gpu_set_array<unsigned int>(unsigned int *,unsigned int, int, int);
template bool gpu_set_array<int2>(int2 *,int2, int, int);
template bool gpu_set_array<scalar>(scalar *,scalar, int, int);
template bool gpu_set_array<dVec>(dVec *,dVec, int, int);
template bool gpu_set_array<cubicLatticeDerivativeVector>(cubicLatticeDerivativeVector *,cubicLatticeDerivativeVector, int, int);
/** @} */ //end of group declaration
