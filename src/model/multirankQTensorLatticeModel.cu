#include "hip/hip_runtime.h"
#include "multirankQTensorLatticeModel.cuh"
/*! \file multirankQTensorLatticeModel.cu */
/*!
    \addtogroup modelKernels
    @{
*/

__device__ int inferDirectionFromIndex(int i,int3 latticeSites, int &startIdx)
    {
    startIdx = 0;
    int base = latticeSites.y*latticeSites.z;
    if (i < base)
        return 0;
    startIdx = base;
    base += latticeSites.y*latticeSites.z;
    if (i < base)
        return 1;
    startIdx = base;
    base += latticeSites.x*latticeSites.z;
    if (i < base)
        return 2;
    startIdx = base;
    base += latticeSites.x*latticeSites.z;
    if (i < base)
        return 3;
    startIdx = base;
    base += latticeSites.x*latticeSites.y;
    if (i < base)
        return 4;
    startIdx = base;
    base += latticeSites.x*latticeSites.y;
    if (i < base)
        return 5;

    startIdx = base;
    base += latticeSites.z;
    if (i < base)
        return 6;
    startIdx = base;
    base += latticeSites.z;
    if (i < base)
        return 7;
    startIdx = base;
    base += latticeSites.y;
    if (i < base)
        return 8;
    startIdx = base;
    base += latticeSites.y;
    if (i < base)
        return 9;
    startIdx = base;
    base += latticeSites.z;
    if (i < base)
        return 10;
    startIdx = base;
    base += latticeSites.z;
    if (i < base)
        return 11;
    startIdx = base;
    base += latticeSites.y;
    if (i < base)
        return 12;
    startIdx = base;
    base += latticeSites.y;
    if (i < base)
        return 13;
    startIdx = base;
    base += latticeSites.x;
    if (i < base)
        return 14;
    startIdx = base;
    base += latticeSites.x;
    if (i < base)
        return 15;
    startIdx = base;
    base += latticeSites.x;
    if (i < base)
        return 16;
    startIdx = base;
    base += latticeSites.x;
    if (i < base)
        return 17;

    startIdx = base;

    return 18 + i-base;
    }

__device__ void getBufferInt3(int idx, int3 &pos,int directionType,int startIndex,int3 latticeSites)
    {
    int index = idx - startIndex;
    switch(directionType)
        {
        case 0:
            pos.z = index / latticeSites.y; pos.y = index % latticeSites.y;
            pos.x = 0;
            break;
        case 1:
            pos.z = index / latticeSites.y; pos.y = index % latticeSites.y;
            pos.x = latticeSites.x-1;
            break;
        case 2:
            pos.z = index / latticeSites.x; pos.x = index % latticeSites.x;
            pos.y =  0;
            break;
        case 3:
            pos.z = index / latticeSites.x; pos.x = index % latticeSites.x;
            pos.y =  latticeSites.y-1;
            break;
        case 4:
            pos.y = index / latticeSites.x; pos.x = index % latticeSites.x;
            pos.z =  0;
            break;
        case 5:
            pos.y = index / latticeSites.x; pos.x = index % latticeSites.x;
            pos.z =  latticeSites.z-1;
            break;
        //edges
        case 6:
            pos.x =  0;
            pos.y =  0;
            pos.z = index;
            break;
        case 7:
            pos.x =  0;
            pos.y =  latticeSites.y-1;
            pos.z = index;
            break;
        case 8:
            pos.x =  0;
            pos.z =  0;
            pos.y = index;
            break;
        case 9:
            pos.x =  0 ;
            pos.z =  latticeSites.z-1;
            pos.y = index;
            break;
        case 10:
            pos.x =  latticeSites.x-1;
            pos.y =  0;
            pos.z = index;
            break;
        case 11:
            pos.x =  latticeSites.x-1;
            pos.y =  latticeSites.y-1;
            pos.z = index;
            break;
        case 12:
            pos.x =  latticeSites.x-1;
            pos.z =  0;
            pos.y = index;
            break;
        case 13:
            pos.x =  latticeSites.x-1;
            pos.z =  latticeSites.z-1;
            pos.y = index;
            break;
        case 14:
            pos.y =  0;
            pos.z =  0;
            pos.x = index;
            break;
        case 15:
            pos.y =  0;
            pos.z =  latticeSites.z-1;
            pos.x = index;
            break;
        case 16:
            pos.y =  latticeSites.y-1;
            pos.z =  0;
            pos.x = index;
            break;
        case 17:
            pos.y =  latticeSites.y-1;
            pos.z =  latticeSites.z-1;
            pos.x = index;
            break;
        //corners
        case 18:
            pos.x =  0;
            pos.y =  0;
            pos.z =  0;
            break;
        case 19:
            pos.x =  0;
            pos.y =  0;
            pos.z =  latticeSites.z-1;
            break;
        case 20:
            pos.x =  0 ;
            pos.y =  latticeSites.y-1;
            pos.z =  0;
            break;
        case 21:
            pos.x =  0 ;
            pos.y =  latticeSites.y-1 ;
            pos.z =  latticeSites.z-1 ;
            break;
        case 22:
            pos.x =  latticeSites.x-1 ;
            pos.y =  0;
            pos.z =  0;
            break;
        case 23:
            pos.x =  latticeSites.x-1;
            pos.y =  0 ;
            pos.z =  latticeSites.z-1;
            break;
        case 24:
            pos.x =  latticeSites.x-1;
            pos.y =  latticeSites.y-1;
            pos.z =  0 ;
            break;
        case 25:
            pos.x =  latticeSites.x-1;
            pos.y =  latticeSites.y-1;
            pos.z =  latticeSites.z-1;
            break;
        }
    }

__global__ void gpu_prepareSendingBuffer_kernel(int *type,
                               dVec *position,
                               int *iBuf,
                               scalar *dBuf,
                               int3 latticeSites,
                               Index3D latticeIndex,
                               int maxIndex)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx >= maxIndex)
        return;

    int startIndex;
    int directionType = inferDirectionFromIndex(idx,latticeSites,startIndex);
    int3 pos;
    getBufferInt3(idx,pos,directionType,startIndex,latticeSites);
    int currentSite = latticeIndex(pos);


    iBuf[idx] = type[currentSite];
    for (int dd = 0; dd < DIMENSION; ++dd)
        dBuf[DIMENSION*idx+dd] = position[currentSite][dd];
    };

bool gpu_prepareSendingBuffer(int *type,
                            dVec *position,
                            int *iBuf,
                            scalar *dBuf,
                            int3 latticeSites,
                            Index3D latticeIndex,
                            int maxIndex,
                            int blockSize)
    {
    int block_size = blockSize;

    if (maxIndex < 128) block_size = 16;
    unsigned int nblocks  = maxIndex/block_size + 1;

    gpu_prepareSendingBuffer_kernel<<<nblocks,block_size>>>(type,position,iBuf,dBuf,latticeSites,latticeIndex,maxIndex);

    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

__global__ void gpu_copyReceivingBuffer_kernel(int *type,
                               dVec *position,
                               int *iBuf,
                               scalar *dBuf,
                               int N,
                               int maxIndex)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx >= maxIndex)
        return;
    int currentSite = idx + N;
    type[currentSite] = iBuf[idx];
    for (int dd = 0; dd < DIMENSION; ++dd)
        position[currentSite][dd] = dBuf[DIMENSION*idx+dd];
    };

bool gpu_copyReceivingBuffer(int *type,
                            dVec *position,
                            int *iBuf,
                            scalar *dBuf,
                            int N,
                            int maxIndex,
                            int blockSize)
    {
    int block_size = blockSize;

    if (maxIndex < 128) block_size = 16;
    unsigned int nblocks  = maxIndex/block_size + 1;

    gpu_copyReceivingBuffer_kernel<<<nblocks,block_size>>>(type,position,iBuf,dBuf,N,maxIndex);

    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration
