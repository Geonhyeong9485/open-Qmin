#include "hip/hip_runtime.h"
#include "multirankQTensorLatticeModel.cuh"
/*! \file multirankQTensorLatticeModel.cu */
/*!
    \addtogroup modelKernels
    @{
*/

__device__ int deviceIndexInExpandedDataArray(int3 &pos,int3 &latticeSites, int3 &expandedLatticeSites, Index3D &latticeIndex,bool xHalo, bool yHalo, bool zHalo )
    {
    if(pos.x <0 && !xHalo)
        pos.x = latticeSites.x-1;
    if(pos.x ==latticeSites.x && !xHalo)
        pos.x = 0;
    if(pos.y <0 && !yHalo)
        pos.y = latticeSites.y-1;
    if(pos.y ==latticeSites.y && !yHalo)
        pos.y = 0;
    if(pos.z <0 && !zHalo)
        pos.z = latticeSites.z-1;
    if(pos.z ==latticeSites.z && !zHalo)
        pos.z = 0;

    if(pos.x < latticeSites.x && pos.y < latticeSites.y && pos.z < latticeSites.z && pos.x >=0 && pos.y >= 0 && pos.z >= 0)
        return latticeIndex(pos);

    //next consider the x = -1 face (total Ly * total Lz)
    int base = latticeIndex.getNumElements();
    if(pos.x == -1)
        return base + pos.y + expandedLatticeSites.y*pos.z;
    //next the x + latticeSites.x + 1 face (note the off-by one fenceposting)
    base +=expandedLatticeSites.y*expandedLatticeSites.z;
    if(pos.x == latticeSites.x)
        return base + pos.y + expandedLatticeSites.y*pos.z;
    base +=expandedLatticeSites.y*expandedLatticeSites.z;
    //next consider the y = -1 face...  0 <=x < latticeSites, by -1 <= z <= latticeSites.z+1
    if(pos.y == -1)
        return base + pos.x + latticeSites.x*pos.z;
    base +=latticeSites.x*expandedLatticeSites.z;
    if(pos.y == latticeSites.y)
        return base + pos.x + latticeSites.x*pos.z;
    base +=latticeSites.x*expandedLatticeSites.z;

    //finally, the z-faces, for which x and y can only be 0 <= letter < latticeSites
    if(pos.z == -1)
        return base + pos.x + latticeSites.x*pos.y;
    base +=latticeSites.x*latticeSites.y;
    if(pos.z == latticeSites.z)
        return base + pos.x + latticeSites.x*pos.y;

    return -1;
    };

__global__ void gpu_mrqtlm_data_to_buffer_kernel(int *type,
                               dVec *position,
                               int *iBuf,
                               scalar *dBuf,
                               int size1start,
                               int size1end,
                               int size2start,
                               int size2end,
                               int xyz,
                               int plane,
                               int3 latticeSites,
                               int3 expandedLatticeSites,
                               Index3D latticeIndex,
                               bool xHalo,
                               bool yHalo,
                               bool zHalo)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
//    int size1 = size1end-size1start;
    int size2 = size2end-size2start;
    //idx = ii * size2 + jj, with possible offsets if the start is negative
    int jj = (idx % size2) + size2start;
    int ii = (idx / size2) + size1start;
    int3 lPos;
    if(xyz ==0)
        {lPos.x = plane; lPos.y = ii; lPos.z = jj;}
    else if(xyz ==1)
        {lPos.x = ii; lPos.y = plane; lPos.z = jj;}
    else if (xyz ==2)
        {lPos.x = ii; lPos.y = jj; lPos.z = plane;}
    int currentSite = deviceIndexInExpandedDataArray(lPos,latticeSites,expandedLatticeSites,latticeIndex,xHalo,yHalo,zHalo);
        
    iBuf[idx] = type[currentSite];
    for(int dd = 0; dd < DIMENSION; ++dd)
        dBuf[DIMENSION*idx+dd] = position[currentSite][dd];
    };

__global__ void gpu_mrqtlm_buffer_to_data_kernel(int *type,
                               dVec *position,
                               int *iBuf,
                               scalar *dBuf,
                               int size1start,
                               int size1end,
                               int size2start,
                               int size2end,
                               int xyz,
                               int plane,
                               int3 latticeSites,
                               int3 expandedLatticeSites,
                               Index3D latticeIndex,
                               bool xHalo,
                               bool yHalo,
                               bool zHalo)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
//    int size1 = size1end-size1start;
    int size2 = size2end-size2start;
    int size1 = size1end-size1start;
    if(idx >=size1*size2)
        return;
    //idx = ii * size2 + jj, with possible offsets if the start is negative
    int jj = (idx % size2) + size2start;
    int ii = (idx / size2) + size1start;
    int3 lPos;
    if(xyz ==0)
        {lPos.x = plane; lPos.y = ii; lPos.z = jj;}
    else if(xyz ==1)
        {lPos.x = ii; lPos.y = plane; lPos.z = jj;}
    else if (xyz ==2)
        {lPos.x = ii; lPos.y = jj; lPos.z = plane;}
    int currentSite = deviceIndexInExpandedDataArray(lPos,latticeSites,expandedLatticeSites,latticeIndex,xHalo,yHalo,zHalo);
        
    type[currentSite] = iBuf[idx];
    for(int dd = 0; dd < DIMENSION; ++dd)
        position[currentSite][dd] = dBuf[DIMENSION*idx+dd];
    };

bool gpu_mrqtlm_buffer_data_exchange(bool sending,
                               int *type,
                               dVec *position,
                               int *iBuf,
                               scalar *dBuf,
                               int size1start,
                               int size1end,
                               int size2start,
                               int size2end,
                               int xyz,
                               int plane,
                               int3 latticeSites,
                               int3 expandedLatticeSites,
                               Index3D &latticeIndex,
                               bool xHalo,
                               bool yHalo,
                               bool zHalo,
                               int blockSize)
    {
    int block_size = blockSize;
    int Size1 = size1end-size1start;
    int Size2 = size2end-size2start;
    int N = Size1*Size2;
    if (N < 128) block_size = 16;
    unsigned int nblocks  = N/block_size + 1;
    
    if(sending)
        {
        gpu_mrqtlm_data_to_buffer_kernel<<<nblocks,block_size>>>(type,position,iBuf,dBuf,size1start,size1end,size2start,size2end,
            xyz,plane,latticeSites,expandedLatticeSites,latticeIndex,xHalo,yHalo,zHalo);
        }
    else
        {
        gpu_mrqtlm_buffer_to_data_kernel<<<nblocks,block_size>>>(type,position,iBuf,dBuf,size1start,size1end,size2start,size2end,
            xyz,plane,latticeSites,expandedLatticeSites,latticeIndex,xHalo,yHalo,zHalo);
        };

    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };



/** @} */ //end of group declaration
