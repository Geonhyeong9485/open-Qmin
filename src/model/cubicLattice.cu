#include "hip/hip_runtime.h"
#include "cubicLattice.cuh"
#include "functions.h"
/*! \file cubicLattice.cu */

/*!
    \addtogroup modelKernels
    @{
*/
__global__ void gpu_set_random_spins_kernel(dVec *pos, hiprandState *rngs,int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    hiprandState randState;
    randState = rngs[blockIdx.x];
    for (int j =0 ; j < threadIdx.x; ++j)
        hiprand(&randState);
    for (int dd = 0; dd < DIMENSION; ++dd)
        pos[idx][dd] = hiprand_normal(&randState);
    scalar lambda = sqrt(dot(pos[idx],pos[idx]));
    pos[idx] = (1/lambda)*pos[idx];
    rngs[blockIdx.x] = randState;
    return;
    };

bool gpu_set_random_spins(dVec *d_pos,
                          hiprandState *rngs,
                          int blockSize,
                          int nBlocks,
                          int N
                          )
    {
    cout << "calling gpu spin setting" << endl;
    gpu_set_random_spins_kernel<<<nBlocks,blockSize>>>(d_pos,rngs,N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

__global__ void gpu_update_spins_kernel(dVec *d_disp,
                      dVec *d_pos,
                      scalar scale,
                      int N,
                      bool normalize)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    d_pos[idx] += scale*d_disp[idx];
    if(normalize)
        {
        scalar nrm =norm(d_pos[idx]);
        d_pos[idx] = (1.0/nrm)*d_pos[idx];
        }
    }

__global__ void gpu_update_spins_simple_kernel(dVec *d_disp,
                      dVec *d_pos,
                      int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int pidx = idx/DIMENSION;
    if(pidx>=N) return;
    int didx = idx%DIMENSION;

    d_pos[pidx][didx] += d_disp[pidx][didx];
    }

bool gpu_update_spins(dVec *d_disp,
                      dVec *d_pos,
                      scalar scale,
                      int N,
                      bool normalize)
    {
    unsigned int block_size = 1024;
    if (N < 128) block_size = 16;
    unsigned int nblocks  = N/block_size + 1;
    if(!normalize && scale == 1.)
        {
        nblocks = DIMENSION*N/block_size + 1;
        gpu_update_spins_simple_kernel<<<nblocks,block_size>>>(d_disp,d_pos,N);
        }
    else
        gpu_update_spins_kernel<<<nblocks,block_size>>>(d_disp,d_pos,scale,N,normalize);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

/** @} */ //end of group declaration
