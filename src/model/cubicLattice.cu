#include "hip/hip_runtime.h"
#include "cubicLattice.cuh"
#include "functions.h"
/*! \file cubicLattice.cu */

/*!
    \addtogroup utilityKernels
    @{
*/
__global__ void gpu_set_random_spins_kernel(dVec *pos, hiprandState *rngs,int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    hiprandState randState;
    randState = rngs[blockIdx.x];
    for (int j =0 ; j < threadIdx.x; ++j)
        hiprand(&randState);
    for (int dd = 0; dd < DIMENSION; ++dd)
        pos[idx][dd] = hiprand_normal(&randState);
    scalar lambda = sqrt(dot(pos[idx],pos[idx]));
    pos[idx] = (1/lambda)*pos[idx];
    rngs[blockIdx.x] = randState;
    return;
    };

bool gpu_set_random_spins(dVec *d_pos,
                          hiprandState *rngs,
                          int blockSize,
                          int nBlocks,
                          int N
                          )
    {
    cout << "calling gpu spin setting" << endl;
    gpu_set_random_spins_kernel<<<nBlocks,blockSize>>>(d_pos,rngs,N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

__global__ void gpu_set_random_nematic_qTensors_kernel(dVec *pos, int *type, hiprandState *rngs,scalar amplitude, bool globallyAligned, scalar globalTheta, scalar globalPhi,int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    hiprandState randState;
    randState = rngs[idx];

    scalar theta = acos(2.0*hiprand_uniform(&randState)-1);
    scalar phi = 2.0*PI*hiprand_uniform(&randState);
    if(globallyAligned)
        {
        theta = globalTheta;
        phi = globalPhi;
        }
    if(type[idx] <=0)
        {
        pos[idx][0] = amplitude*(sin(theta)*sin(theta)*cos(phi)*cos(phi)-1.0/3.0);
        pos[idx][1] = amplitude*sin(theta)*sin(theta)*cos(phi)*sin(phi);
        pos[idx][2] = amplitude*sin(theta)*cos(theta)*cos(phi);
        pos[idx][3] = amplitude*(sin(theta)*sin(theta)*sin(phi)*sin(phi)-1.0/3.0);
        pos[idx][4] = amplitude*sin(theta)*cos(theta)*sin(phi);
        }
    rngs[idx] = randState;
    return;
    };

bool gpu_set_random_nematic_qTensors(dVec *d_pos,
                          int *d_types,
                          hiprandState *rngs,
                          scalar amplitude,
                          int blockSize,
                          int nBlocks,
                          bool globallyAligned,
                          scalar theta,
                          scalar phi,
                          int N
                          )
    {
    if(DIMENSION <5)
        {
        printf("\nAttempting to initialize Q-tensors with incorrectly set dimension...change the root CMakeLists.txt file to have dimension 5 and recompile\n");
        throw std::exception();
        }
    gpu_set_random_nematic_qTensors_kernel<<<nBlocks,blockSize>>>(d_pos,d_types, rngs,amplitude, globallyAligned, theta, phi,N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

__global__ void gpu_update_spins_kernel(dVec *d_disp,
                      dVec *d_pos,
                      scalar scale,
                      int N,
                      bool normalize)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    d_pos[idx] += scale*d_disp[idx];
    if(normalize)
        {
        scalar nrm =norm(d_pos[idx]);
        d_pos[idx] = (1.0/nrm)*d_pos[idx];
        }
    }

__global__ void gpu_update_spins_simple_kernel(dVec *d_disp,
                      dVec *d_pos,
                      int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int pidx = idx/DIMENSION;
    if(pidx>=N) return;
    int didx = idx%DIMENSION;

    d_pos[pidx][didx] += d_disp[pidx][didx];
    }

bool gpu_update_spins(dVec *d_disp,
                      dVec *d_pos,
                      scalar scale,
                      int N,
                      bool normalize)
    {
    unsigned int block_size = 1024;
    if (N < 128) block_size = 16;
    unsigned int nblocks  = N/block_size + 1;
    if(!normalize && scale == 1.)
        {
        nblocks = DIMENSION*N/block_size + 1;
        gpu_update_spins_simple_kernel<<<nblocks,block_size>>>(d_disp,d_pos,N);
        }
    else
        gpu_update_spins_kernel<<<nblocks,block_size>>>(d_disp,d_pos,scale,N,normalize);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

/** @} */ //end of group declaration
